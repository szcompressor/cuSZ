/**
 * @file gather_scatter.cu
 * @author Jiannan Tian
 * @brief Gather/scatter method to handle cuSZ prediction outlier.
 * @version 0.1
 * @date 2020-09-20
 * Created on 2020-09-10
 *
 * @copyright (C) 2020 by Washington State University, Argonne National Laboratory
 * See LICENSE in top-level directory
 *
 */

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include <cassert>
#include <iostream>
using std::cout;
using std::endl;

#include "cuda_error_handling.cuh"
#include "format.hh"
#include "gather_scatter.cuh"
#include "io.hh"

using handle_t = hipsparseHandle_t;
using stream_t = hipStream_t;
using descr_t  = hipsparseMatDescr_t;

template <typename DType>
void cusz::impl::GatherAsCSR(DType* d_A, size_t lenA, size_t ldA, size_t m, size_t n, int* nnz, std::string* fo)
{
    uint8_t* outbin;
    size_t   lrp, lci, lv, ltotal;

    {
        handle_t handle        = nullptr;
        stream_t stream        = nullptr;
        descr_t  descr         = nullptr;
        int*     d_nnz_per_row = nullptr;
        int*     d_row_ptr     = nullptr;
        int*     d_col_ind     = nullptr;
        DType*   d_csr_val     = nullptr;

        // clang-format off
        CHECK_CUDA(hipStreamCreateWithFlags   ( &stream, hipStreamNonBlocking        )); // 1. create stream
        CHECK_CUSPARSE(hipsparseCreate          ( &handle                               )); // 2. create handle
        CHECK_CUSPARSE(hipsparseSetStream       (  handle, stream                       )); // 3. bind stream
        CHECK_CUSPARSE(hipsparseCreateMatDescr  ( &descr                                )); // 4. create descr
        CHECK_CUSPARSE(hipsparseSetMatIndexBase (  descr,  HIPSPARSE_INDEX_BASE_ZERO     )); // zero based
        CHECK_CUSPARSE(hipsparseSetMatType      (  descr,  HIPSPARSE_MATRIX_TYPE_GENERAL )); // type

        CHECK_CUDA(hipMalloc((void**)&d_nnz_per_row, sizeof(int) * m));

        CHECK_CUSPARSE(hipsparseSnnz(
            handle, HIPSPARSE_DIRECTION_ROW, // parsed by row
            m, n, descr, d_A, ldA,          // descrption of d_A
            d_nnz_per_row, nnz)             // output
        );

        lrp    = sizeof(int)   * (m + 1);
        lci    = sizeof(int)   * *nnz;
        lv     = sizeof(DType) * *nnz;
        ltotal = lrp + lci + lv;
        outbin = new uint8_t[ltotal];
        CHECK_CUDA(hipMalloc((void**)&d_row_ptr, lrp));
        CHECK_CUDA(hipMalloc((void**)&d_col_ind, lci));
        CHECK_CUDA(hipMalloc((void**)&d_csr_val, lv ));

        CHECK_CUSPARSE(hipsparseSdense2csr(
            handle,                             //
            m, n, descr, d_A, ldA,              // descritpion of d_A
            d_nnz_per_row,                      // prefileld by nnz() func
            d_csr_val, d_row_ptr, d_col_ind)    // output
        );
        CHECK_CUDA(hipDeviceSynchronize());

        CHECK_CUDA(hipMemcpy(outbin,             d_row_ptr, lrp, hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(outbin + lrp,       d_col_ind, lci, hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(outbin + lrp + lci, d_csr_val, lv,  hipMemcpyDeviceToHost));

        if (d_row_ptr)  hipFree(d_row_ptr);
        if (d_col_ind)  hipFree(d_col_ind);
        if (d_csr_val)  hipFree(d_csr_val);
        if (d_nnz_per_row) hipFree(d_nnz_per_row);
        if (handle)     hipsparseDestroy(handle);
        if (stream)     hipStreamDestroy(stream);
        if (descr)      hipsparseDestroyMatDescr(descr);
        // clang-format on
    }

    // cout << log_dbg << "outlier_bin byte length:\t" << ltotal << endl;
    io::WriteArrayToBinary(*fo, outbin, ltotal);
    delete[] outbin;
};

template void
cusz::impl::GatherAsCSR<float>(float* d_A, size_t lenA, size_t ldA, size_t m, size_t n, int* nnz, std::string* fo);

template <typename DType>
void cusz::impl::ScatterFromCSR(DType* d_A, size_t lenA, size_t ldA, size_t m, size_t n, int* nnz, std::string* fi)
{
    // clang-format off
    auto lrp         = sizeof(int) * (ldA + 1);
    auto lci         = sizeof(int) * *nnz;
    auto lv          = sizeof(DType) * *nnz;
    auto l_total     = lrp + lci + lv;
    auto outlier_bin = io::ReadBinaryFile<uint8_t>(*fi, l_total);
    auto row_ptr     = reinterpret_cast<int*>(outlier_bin);
    auto col_ind     = reinterpret_cast<int*>(outlier_bin + lrp);
    auto csr_val     = reinterpret_cast<DType*>(outlier_bin + lrp + lci);  // TODO template
    // clang-format on

    {
        handle_t handle    = nullptr;
        stream_t stream    = nullptr;
        descr_t  descr     = nullptr;
        int*     d_row_ptr = nullptr;
        int*     d_col_ind = nullptr;
        DType*   d_csr_val = nullptr;

        // clang-format off
        CHECK_CUDA(hipStreamCreateWithFlags   ( &stream, hipStreamNonBlocking        )); // 1. create stream
        CHECK_CUSPARSE(hipsparseCreate          ( &handle                               )); // 2. create handle
        CHECK_CUSPARSE(hipsparseSetStream       (  handle, stream                       )); // 3. bind stream
        CHECK_CUSPARSE(hipsparseCreateMatDescr  ( &descr                                )); // 4. create descr
        CHECK_CUSPARSE(hipsparseSetMatIndexBase (  descr,  HIPSPARSE_INDEX_BASE_ZERO     )); // zero based
        CHECK_CUSPARSE(hipsparseSetMatType      (  descr,  HIPSPARSE_MATRIX_TYPE_GENERAL )); // type

        CHECK_CUDA(hipMalloc( (void**)&d_row_ptr,   lrp ));
        CHECK_CUDA(hipMalloc( (void**)&d_col_ind,   lci ));
        CHECK_CUDA(hipMalloc( (void**)&d_csr_val,      lv    ));
        CHECK_CUDA(hipMemcpy( d_row_ptr, row_ptr, lrp, hipMemcpyHostToDevice ));
        CHECK_CUDA(hipMemcpy( d_col_ind, col_ind, lci, hipMemcpyHostToDevice ));
        CHECK_CUDA(hipMemcpy( d_csr_val, csr_val, lv,  hipMemcpyHostToDevice ));

        CHECK_CUSPARSE(hipsparseScsr2dense(handle, m, n, descr, d_csr_val, d_row_ptr, d_col_ind, d_A, ldA));
        CHECK_CUDA(hipDeviceSynchronize());

        if (d_row_ptr) hipFree(d_row_ptr);
        if (d_col_ind) hipFree(d_col_ind);
        if (d_csr_val) hipFree(d_csr_val);
        if (handle)    hipsparseDestroy(handle);
        if (stream)    hipStreamDestroy(stream);
        if (descr)     hipsparseDestroyMatDescr(descr);
        // clang-format on
    }

    logall(log_info, "load outlier");

    delete[] outlier_bin;
}

template void
cusz::impl::ScatterFromCSR<float>(float* d_A, size_t lenA, size_t ldA, size_t m, size_t n, int* nnz, std::string* fi);

void cusz::impl::PruneGatherAsCSR(
    float*       d_A,  //
    size_t       lenA,
    const int    lda,
    const int    m,
    const int    n,
    int&         nnzC,
    std::string* fo)
{
    handle_t handle       = nullptr;
    stream_t stream       = nullptr;
    descr_t  descr        = nullptr;
    int*     d_row_ptr    = nullptr;
    int*     d_col_ind    = nullptr;
    float*   d_csr_val    = nullptr;
    size_t   lworkInBytes = 0;
    char*    d_work       = nullptr;
    float    threshold    = 0;

    // clang-format off
    CHECK_CUDA(hipStreamCreateWithFlags   ( &stream, hipStreamNonBlocking        )); // 1. create stream
    CHECK_CUSPARSE(hipsparseCreate          ( &handle                               )); // 2. create handle
    CHECK_CUSPARSE(hipsparseSetStream       (  handle, stream                       )); // 3. bind stream
    CHECK_CUSPARSE(hipsparseCreateMatDescr  ( &descr                                )); // 4. create descr
    CHECK_CUSPARSE(hipsparseSetMatIndexBase (  descr,  HIPSPARSE_INDEX_BASE_ZERO     )); // zero based
    CHECK_CUSPARSE(hipsparseSetMatType      (  descr,  HIPSPARSE_MATRIX_TYPE_GENERAL )); // type
    // clang-format on

    CHECK_CUDA(hipMalloc((void**)&d_row_ptr, sizeof(int) * (m + 1)));

    CHECK_CUSPARSE(hipsparseSpruneDense2csr_bufferSizeExt(  //
        handle, m, n, d_A, lda, &threshold, descr, d_csr_val, d_row_ptr, d_col_ind, &lworkInBytes));

    // printf("lworkInBytes (prune) = %lld \n", (long long)lworkInBytes);
    if (nullptr != d_work) hipFree(d_work);

    CHECK_CUDA(hipMalloc((void**)&d_work, lworkInBytes));

    /* step 4: compute row_ptrC and nnzC */
    CHECK_CUSPARSE(hipsparseSpruneDense2csrNnz(  //
        handle, m, n, d_A, lda, &threshold, descr, d_row_ptr, &nnzC, d_work));
    CHECK_CUDA(hipDeviceSynchronize());

    if (0 == nnzC) cout << log_info << "No outlier." << endl;

    /* step 5: compute col_indC and csr_valC */
    CHECK_CUDA(hipMalloc((void**)&d_col_ind, sizeof(int) * nnzC));
    CHECK_CUDA(hipMalloc((void**)&d_csr_val, sizeof(float) * nnzC));

    CHECK_CUSPARSE(hipsparseSpruneDense2csr(  //
        handle, m, n, d_A, lda, &threshold, descr, d_csr_val, d_row_ptr, d_col_ind, d_work));
    CHECK_CUDA(hipDeviceSynchronize());

    /* step 6: output C */
    auto lrp    = sizeof(int) * (m + 1);
    auto lci    = sizeof(int) * nnzC;
    auto lv     = sizeof(float) * nnzC;
    auto ltotal = lrp + lci + lv;
    auto outbin = new uint8_t[ltotal];

    // clang-format off
    CHECK_CUDA(hipMemcpy(outbin,             d_row_ptr, lrp, hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(outbin + lrp,       d_col_ind, lci, hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(outbin + lrp + lci, d_csr_val, lv,  hipMemcpyDeviceToHost));
    // clang-format on

    io::WriteArrayToBinary(*fo, outbin, ltotal);

    if (d_row_ptr) hipFree(d_row_ptr);
    if (d_col_ind) hipFree(d_col_ind);
    if (d_csr_val) hipFree(d_csr_val);
    if (handle) hipsparseDestroy(handle);
    if (stream) hipStreamDestroy(stream);
    if (descr) hipsparseDestroyMatDescr(descr);
    if (outbin) delete[] outbin;
}
