#include "hip/hip_runtime.h"
/**
 * @file huffman_codec.cu
 * @author Jiannan Tian
 * @brief Wrapper of Huffman codec.
 * @version 0.1
 * @date 2020-09-20
 * Created on 2020-02-02
 *
 * @copyright (C) 2020 by Washington State University, The University of Alabama, Argonne National Laboratory
 * See LICENSE in top-level directory
 *
 */

#include <stddef.h>
#include <stdint.h>
#include <cstdio>
#include <limits>
#include "huffman_codec.cuh"

using uint8__t = uint8_t;

template <typename Q, typename H>
__global__ void EncodeFixedLen(Q* data, H* hcoded, size_t data_len, H* codebook)
{
    size_t gid = blockDim.x * blockIdx.x + threadIdx.x;
    if (gid >= data_len) return;
    hcoded[gid] = codebook[data[gid]];  // try to exploit cache?
    __syncthreads();
}

template <typename Q>
__global__ void Deflate(
    Q*      hcoded,  //
    size_t  len,
    size_t* densely_meta,
    int     PART_SIZE)
{
    size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= (len - 1) / PART_SIZE + 1) return;
    uint8_t bitwidth;
    size_t  densely_coded_lsb_pos = sizeof(Q) * 8, total_bitwidth = 0;
    size_t  ending = (gid + 1) * PART_SIZE <= len ? PART_SIZE : len - gid * PART_SIZE;
    //    if ((gid + 1) * PART_SIZE > len) printf("\n\ngid %lu\tending %lu\n\n", gid, ending);
    Q  msb_bw_word_lsb, _1, _2;
    Q* current = hcoded + gid * PART_SIZE;
    for (size_t i = 0; i < ending; i++) {
        msb_bw_word_lsb = hcoded[gid * PART_SIZE + i];
        bitwidth        = *((uint8_t*)&msb_bw_word_lsb + (sizeof(Q) - 1));

        *((uint8_t*)&msb_bw_word_lsb + sizeof(Q) - 1) = 0x0;
        if (densely_coded_lsb_pos == sizeof(Q) * 8) *current = 0x0;  // a new unit of data type
        if (bitwidth <= densely_coded_lsb_pos) {
            densely_coded_lsb_pos -= bitwidth;
            *current |= msb_bw_word_lsb << densely_coded_lsb_pos;
            if (densely_coded_lsb_pos == 0) {
                densely_coded_lsb_pos = sizeof(Q) * 8;
                ++current;
            }
        }
        else {
            // example: we have 5-bit code 11111 but 3 bits left for (*current)
            // we put first 3 bits of 11111 to the last 3 bits of (*current)
            // and put last 2 bits from MSB of (*(++current))
            // the comment continues with the example
            _1 = msb_bw_word_lsb >> (bitwidth - densely_coded_lsb_pos);
            _2 = msb_bw_word_lsb << (sizeof(Q) * 8 - (bitwidth - densely_coded_lsb_pos));
            *current |= _1;
            *(++current) = 0x0;
            *current |= _2;
            densely_coded_lsb_pos = sizeof(Q) * 8 - (bitwidth - densely_coded_lsb_pos);
        }
        total_bitwidth += bitwidth;
    }
    *(densely_meta + gid) = total_bitwidth;
}

template <typename H, typename T>
__device__ void InflateChunkwise(H* in_huff, T* out_quant, size_t total_bw, uint8_t* singleton)
{
    uint8_t next_bit;
    size_t  idx_bit;
    size_t  idx_byte   = 0;
    size_t  idx_bcoded = 0;
    auto    first      = reinterpret_cast<H*>(singleton);
    auto    entry      = first + sizeof(H) * 8;
    auto    keys       = reinterpret_cast<T*>(singleton + sizeof(H) * (2 * sizeof(H) * 8));
    H       v          = (in_huff[idx_byte] >> (sizeof(H) * 8 - 1)) & 0x1;  // get the first bit
    size_t  l          = 1;
    size_t  i          = 0;
    while (i < total_bw) {
        while (v < first[l]) {  // append next i_cb bit
            ++i;
            idx_byte = i / (sizeof(H) * 8);
            idx_bit  = i % (sizeof(H) * 8);
            next_bit = ((in_huff[idx_byte] >> (sizeof(H) * 8 - 1 - idx_bit)) & 0x1);
            v        = (v << 1) | next_bit;
            ++l;
        }
        out_quant[idx_bcoded++] = keys[entry[l] + v - first[l]];
        {
            ++i;
            idx_byte = i / (sizeof(H) * 8);
            idx_bit  = i % (sizeof(H) * 8);
            next_bit = ((in_huff[idx_byte] >> (sizeof(H) * 8 - 1 - idx_bit)) & 0x1);
            v        = 0x0 | next_bit;
        }
        l = 1;
    }
}

template <typename Q, typename H>
__global__ void Decode(
    H*       densely,     //
    size_t*  dH_meta,     //
    Q*       bcode,       //
    size_t   len,         //
    int      chunk_size,  //
    int      n_chunk,
    uint8_t* singleton,
    size_t   singleton_size)
{
    extern __shared__ uint8_t _s_singleton[];
    if (threadIdx.x == 0) memcpy(_s_singleton, singleton, singleton_size);
    __syncthreads();

    auto dH_bit_meta   = dH_meta;
    auto dH_uInt_entry = dH_meta + n_chunk;

    size_t chunk_id = blockIdx.x * blockDim.x + threadIdx.x;
    // if (chunk_id == 0) printf("n_chunk: %lu\n", n_chunk);
    if (chunk_id >= n_chunk) return;

    InflateChunkwise(                       //
        densely + dH_uInt_entry[chunk_id],  //
        bcode + chunk_size * chunk_id,      //
        dH_bit_meta[chunk_id],              //
        _s_singleton);
    __syncthreads();
};

template __global__ void EncodeFixedLen<uint8__t, uint32_t>(uint8__t*, uint32_t*, size_t, uint32_t*);
template __global__ void EncodeFixedLen<uint8__t, uint64_t>(uint8__t*, uint64_t*, size_t, uint64_t*);
template __global__ void EncodeFixedLen<uint16_t, uint32_t>(uint16_t*, uint32_t*, size_t, uint32_t*);
template __global__ void EncodeFixedLen<uint16_t, uint64_t>(uint16_t*, uint64_t*, size_t, uint64_t*);
template __global__ void EncodeFixedLen<uint32_t, uint32_t>(uint32_t*, uint32_t*, size_t, uint32_t*);
template __global__ void EncodeFixedLen<uint32_t, uint64_t>(uint32_t*, uint64_t*, size_t, uint64_t*);

template __global__ void Deflate<uint32_t>(uint32_t* hcoded, size_t len, size_t* densely_meta, int PART_SIZE);
template __global__ void Deflate<uint64_t>(uint64_t* hcoded, size_t len, size_t* densely_meta, int PART_SIZE);

// H for Huffman, uint{32,64}_t
// T for quant code, uint{8,16,32}_t
template __device__ void InflateChunkwise<uint32_t, uint8__t>(uint32_t*, uint8__t*, size_t, uint8__t*);
template __device__ void InflateChunkwise<uint32_t, uint16_t>(uint32_t*, uint16_t*, size_t, uint8__t*);
template __device__ void InflateChunkwise<uint32_t, uint32_t>(uint32_t*, uint32_t*, size_t, uint8__t*);
template __device__ void InflateChunkwise<uint64_t, uint8__t>(uint64_t*, uint8__t*, size_t, uint8__t*);
template __device__ void InflateChunkwise<uint64_t, uint16_t>(uint64_t*, uint16_t*, size_t, uint8__t*);
template __device__ void InflateChunkwise<uint64_t, uint32_t>(uint64_t*, uint32_t*, size_t, uint8__t*);

template __global__ void Decode<uint8__t, uint32_t>(uint32_t*, size_t*, uint8__t*, size_t, int, int, uint8__t*, size_t);
template __global__ void Decode<uint8__t, uint64_t>(uint64_t*, size_t*, uint8__t*, size_t, int, int, uint8__t*, size_t);
template __global__ void Decode<uint16_t, uint32_t>(uint32_t*, size_t*, uint16_t*, size_t, int, int, uint8__t*, size_t);
template __global__ void Decode<uint16_t, uint64_t>(uint64_t*, size_t*, uint16_t*, size_t, int, int, uint8__t*, size_t);
template __global__ void Decode<uint32_t, uint32_t>(uint32_t*, size_t*, uint32_t*, size_t, int, int, uint8__t*, size_t);
template __global__ void Decode<uint32_t, uint64_t>(uint64_t*, size_t*, uint32_t*, size_t, int, int, uint8__t*, size_t);
