#include "hip/hip_runtime.h"
/**
 * @file cuda_mem.cu
 * @author Jiannan Tian
 * @brief CUDA memory operation wrappers.
 * @version 0.1
 * @date 2020-09-20
 * Created on 2020-04-30
 *
 * @copyright (C) 2020 by Washington State University, The University of Alabama, Argonne National Laboratory
 * See LICENSE in top-level directory
 *
 */

#include <hip/hip_runtime.h>
#include <cassert>
#include <cstddef>
#include <cstdint>
#include "cuda_mem.cuh"

template <typename T>
inline T* mem::CreateCUDASpace(size_t l, uint8_t i)
{
    T* d_var;
    hipMalloc(&d_var, l * sizeof(T));
    hipMemset(d_var, i, l * sizeof(T));
    return d_var;
}

// enum MemcpyDirection { h2d, d2h };

template <typename T>
void mem::CopyBetweenSpaces(T* src, T* dst, size_t l, MemcpyDirection direct)
{
    assert(src != nullptr);
    assert(dst != nullptr);
    if (direct == h2d) { hipMemcpy(dst, src, sizeof(T) * l, hipMemcpyHostToDevice); }
    else if (direct == d2h) {
        hipMemcpy(dst, src, sizeof(T) * l, hipMemcpyDeviceToHost);
    }
    else {
        // TODO log
        exit(1);
    }
}

template <typename T>
inline T* mem::CreateDeviceSpaceAndMemcpyFromHost(T* var, size_t l)
{
    T* d_var;
    hipMalloc(&d_var, l * sizeof(T));
    hipMemcpy(d_var, var, l * sizeof(T), hipMemcpyHostToDevice);
    return d_var;
}
template <typename T>
inline T* mem::CreateHostSpaceAndMemcpyFromDevice(T* d_var, size_t l)
{
    auto var = new T[l];
    hipMemcpy(var, d_var, l * sizeof(T), hipMemcpyDeviceToHost);
    return var;
}

template uint8_t*  mem::CreateCUDASpace<uint8_t>(size_t l, uint8_t i);
template uint16_t* mem::CreateCUDASpace<uint16_t>(size_t l, uint8_t i);
template uint32_t* mem::CreateCUDASpace<uint32_t>(size_t l, uint8_t i);
template uint64_t* mem::CreateCUDASpace<uint64_t>(size_t l, uint8_t i);
template int8_t*   mem::CreateCUDASpace<int8_t>(size_t l, uint8_t i);
template int16_t*  mem::CreateCUDASpace<int16_t>(size_t l, uint8_t i);
template int32_t*  mem::CreateCUDASpace<int32_t>(size_t l, uint8_t i);
template int64_t*  mem::CreateCUDASpace<int64_t>(size_t l, uint8_t i);
template float*    mem::CreateCUDASpace<float>(size_t l, uint8_t i);
template double*   mem::CreateCUDASpace<double>(size_t l, uint8_t i);

template int8_t*   mem::CreateDeviceSpaceAndMemcpyFromHost(int8_t* var, size_t l);
template int16_t*  mem::CreateDeviceSpaceAndMemcpyFromHost(int16_t* var, size_t l);
template int32_t*  mem::CreateDeviceSpaceAndMemcpyFromHost(int32_t* var, size_t l);
template int64_t*  mem::CreateDeviceSpaceAndMemcpyFromHost(int64_t* var, size_t l);
template uint8_t*  mem::CreateDeviceSpaceAndMemcpyFromHost(uint8_t* var, size_t l);
template uint16_t* mem::CreateDeviceSpaceAndMemcpyFromHost(uint16_t* var, size_t l);
template uint32_t* mem::CreateDeviceSpaceAndMemcpyFromHost(uint32_t* var, size_t l);
template uint64_t* mem::CreateDeviceSpaceAndMemcpyFromHost(uint64_t* var, size_t l);
template float*    mem::CreateDeviceSpaceAndMemcpyFromHost(float* var, size_t l);
template double*   mem::CreateDeviceSpaceAndMemcpyFromHost(double* var, size_t l);

template int8_t*   mem::CreateHostSpaceAndMemcpyFromDevice(int8_t* d_var, size_t l);
template int16_t*  mem::CreateHostSpaceAndMemcpyFromDevice(int16_t* d_var, size_t l);
template int32_t*  mem::CreateHostSpaceAndMemcpyFromDevice(int32_t* d_var, size_t l);
template int64_t*  mem::CreateHostSpaceAndMemcpyFromDevice(int64_t* d_var, size_t l);
template uint8_t*  mem::CreateHostSpaceAndMemcpyFromDevice(uint8_t* d_var, size_t l);
template uint16_t* mem::CreateHostSpaceAndMemcpyFromDevice(uint16_t* d_var, size_t l);
template uint32_t* mem::CreateHostSpaceAndMemcpyFromDevice(uint32_t* d_var, size_t l);
template uint64_t* mem::CreateHostSpaceAndMemcpyFromDevice(uint64_t* d_var, size_t l);
template float*    mem::CreateHostSpaceAndMemcpyFromDevice(float* d_var, size_t l);
template double*   mem::CreateHostSpaceAndMemcpyFromDevice(double* d_var, size_t l);
