#include "hip/hip_runtime.h"
/**
 * @file huffman_workflow.cu
 * @author Jiannan Tian, Cody Rivera (cjrivera1@crimson.ua.edu)
 * @brief Workflow of Huffman coding.
 * @version 0.1
 * @date 2020-10-24
 * Created on 2020-04-24
 *
 * @copyright (C) 2020 by Washington State University, The University of Alabama, Argonne National Laboratory
 * See LICENSE in top-level directory
 *
 */

#include <hip/hip_runtime.h>

#include <sys/stat.h>
#include <unistd.h>
#include <algorithm>
#include <bitset>
#include <cassert>
#include <cmath>
#include <functional>
#include <iomanip>
#include <iostream>
#include <numeric>
#include <stdexcept>
#include <string>
#include <tuple>
#include <type_traits>
#include <vector>

#include "../kernel/hist.h"
#include "../kernel/huffman_codec.h"
#include "../type_aliasing.hh"
#include "../type_trait.hh"
#include "../types.hh"
#include "../utils/cuda_err.cuh"
#include "../utils/cuda_mem.cuh"
#include "../utils/format.hh"
#include "../utils/io.hh"
#include "../utils/timer.hh"
#include "deprecated_lossless_huffman.h"

#ifdef MODULAR_ELSEWHERE
#include "cascaded.hpp"
#include "nvcomp.hpp"
#endif

#if __cplusplus >= 201703L
#define CONSTEXPR constexpr
#else
#define CONSTEXPR
#endif

#define nworker blockDim.x

template <typename Huff>
__global__ void cusz::CopyHuffmanUintsDenseToSparse(
    Huff*   input_dn,
    Huff*   output_sp,
    size_t* sp_entries,
    size_t* sp_uints,
    size_t  dn_chunk_size)
{
    auto len      = sp_uints[blockIdx.x];
    auto sp_entry = sp_entries[blockIdx.x];
    auto dn_entry = dn_chunk_size * blockIdx.x;

    for (auto i = 0; i < (len + nworker - 1) / nworker; i++) {
        auto _tid = threadIdx.x + i * nworker;
        if (_tid < len) *(output_sp + sp_entry + _tid) = *(input_dn + dn_entry + _tid);
        __syncthreads();
    }
}

template <typename Huff>
void cusz::GatherSpHuffMetadata(
    size_t* _counts,
    size_t* d_sp_bits,
    size_t  nchunk,
    size_t& total_bits,
    size_t& total_uints)
{
    static const size_t Huff_bytes = sizeof(Huff) * 8;

    auto sp_uints = _counts, sp_bits = _counts + nchunk, sp_entries = _counts + nchunk * 2;

    hipMemcpy(sp_bits, d_sp_bits, nchunk * sizeof(size_t), hipMemcpyDeviceToHost);
    memcpy(sp_uints, sp_bits, nchunk * sizeof(size_t));
    for_each(sp_uints, sp_uints + nchunk, [&](size_t& i) { i = (i + Huff_bytes - 1) / Huff_bytes; });
    memcpy(sp_entries + 1, sp_uints, (nchunk - 1) * sizeof(size_t));
    for (auto i = 1; i < nchunk; i++) sp_entries[i] += sp_entries[i - 1];  // inclusive scan

    total_bits  = std::accumulate(sp_bits, sp_bits + nchunk, (size_t)0);
    total_uints = std::accumulate(sp_uints, sp_uints + nchunk, (size_t)0);

    //    auto fmt_enc1 = "Huffman enc: (#) " + std::to_string(nchunk) + " x " + std::to_string(dn_chunk);
    //    auto fmt_enc2 = std::to_string(total_uints) + " " + std::to_string(sizeof(Huff)) + "-byte words or " +
    //                    std::to_string(total_bits) + " bits";
    //    LogAll(log_dbg, fmt_enc1, "=>", fmt_enc2);
}

#ifdef MODULAR_ELSEWHERE
template <typename T>
void draft::UseNvcompZip(T* space, size_t& len)
{
    int*         uncompressed_data;
    const size_t in_bytes = len * sizeof(T);

    hipMalloc(&uncompressed_data, in_bytes);
    hipMemcpy(uncompressed_data, space, in_bytes, hipMemcpyHostToDevice);
    hipStream_t stream;
    hipStreamCreate(&stream);
    // 2 layers RLE, 1 Delta encoding, bitpacking enabled
    nvcomp::CascadedCompressor<int> compressor(uncompressed_data, in_bytes / sizeof(int), 2, 1, true);
    const size_t                    temp_size = compressor.get_temp_size();
    void*                           temp_space;
    hipMalloc(&temp_space, temp_size);
    size_t output_size = compressor.get_max_output_size(temp_space, temp_size);
    void*  output_space;
    hipMalloc(&output_space, output_size);
    compressor.compress_async(temp_space, temp_size, output_space, &output_size, stream);
    hipStreamSynchronize(stream);
    // TODO ad hoc; should use original GPU space
    memset(space, 0x0, len * sizeof(T));
    len = output_size / sizeof(T);
    hipMemcpy(space, output_space, output_size, hipMemcpyDeviceToHost);

    hipFree(uncompressed_data);
    hipFree(temp_space);
    hipFree(output_space);
    hipStreamDestroy(stream);
}

template <typename T>
void draft::UseNvcompUnzip(T** d_space, size_t& len)
{
    hipStream_t stream;
    hipStreamCreate(&stream);

    nvcomp::Decompressor<int> decompressor(*d_space, len * sizeof(T), stream);
    const size_t              temp_size = decompressor.get_temp_size();
    void*                     temp_space;
    hipMalloc(&temp_space, temp_size);

    const size_t output_count = decompressor.get_num_elements();
    int*         output_space;
    hipMalloc((void**)&output_space, output_count * sizeof(int));

    decompressor.decompress_async(temp_space, temp_size, output_space, output_count, stream);

    hipStreamSynchronize(stream);
    hipFree(*d_space);

    *d_space = mem::CreateCUDASpace<T>((unsigned long)(output_count * sizeof(int)));
    hipMemcpy(*d_space, output_space, output_count * sizeof(int), hipMemcpyDeviceToDevice);
    len = output_count * sizeof(int) / sizeof(T);

    hipFree(output_space);

    hipStreamDestroy(stream);
    hipFree(temp_space);
}

#endif

template <typename Quant, typename Huff, typename Data>
std::tuple<size_t, size_t, size_t> lossless::interface::HuffmanEncode(
    string&  basename,
    Quant*   d_input,
    Huff*    d_canon_cb,
    uint8_t* d_reverse_cb,
    size_t   _nbyte,
    size_t   len,
    int      dn_chunk,
    int      dict_size)
{
    static const auto type_bitcount = sizeof(Huff) * 8;  // canonical Huffman; follows H to decide first and entry type

    auto get_Dg = [](size_t problem_size, size_t Db) { return (problem_size + Db - 1) / Db; };

    auto decode_meta = mem::CreateHostSpaceAndMemcpyFromDevice(d_reverse_cb, _nbyte);
    io::WriteArrayToBinary(
        basename + ".canon", reinterpret_cast<uint8_t*>(decode_meta),
        sizeof(Huff) * (2 * type_bitcount) + sizeof(Quant) * dict_size);
    delete[] decode_meta;

    // Huffman space in dense format (full of zeros), fix-length space
    auto d_huff_dn = mem::CreateCUDASpace<Huff>(len + dn_chunk + HuffConfig::Db_encode);  // TODO ad hoc (big) padding
    {
        auto Db = HuffConfig::Db_encode;
        cusz::EncodeFixedLen_cub<Quant, Huff, HuffConfig::enc_sequentiality>
            <<<get_Dg(len, Db), Db / HuffConfig::enc_sequentiality>>>(d_input, d_huff_dn, len, d_canon_cb);
        hipDeviceSynchronize();
    }

    // deflate
    auto nchunk    = (len + dn_chunk - 1) / dn_chunk;
    auto d_sp_bits = mem::CreateCUDASpace<size_t>(nchunk);
    {
        auto Db = HuffConfig::Db_deflate;
        cusz::Deflate<Huff><<<get_Dg(nchunk, Db), Db>>>(d_huff_dn, len, d_sp_bits, dn_chunk);
        hipDeviceSynchronize();
    }

    // gather metadata (without write) before gathering huff as sp on GPU
    auto   _counts    = new size_t[nchunk * 3]();
    size_t total_bits = 0, total_uints = 0;
    cusz::GatherSpHuffMetadata<Huff>(_counts, d_sp_bits, nchunk, total_bits, total_uints);

    // partially gather on GPU and copy back (TODO fully)
    auto huff_sp = new Huff[total_uints]();
    {
        auto d_huff_sp = mem::CreateCUDASpace<Huff>(total_uints);
        auto d_uints   = mem::CreateDeviceSpaceAndMemcpyFromHost(_counts, nchunk);               // sp_uints
        auto d_entries = mem::CreateDeviceSpaceAndMemcpyFromHost(_counts + nchunk * 2, nchunk);  // sp_entries
        cusz::CopyHuffmanUintsDenseToSparse<<<nchunk, 128>>>(d_huff_dn, d_huff_sp, d_entries, d_uints, dn_chunk);
        hipDeviceSynchronize();
        hipMemcpy(huff_sp, d_huff_sp, total_uints * sizeof(Huff), hipMemcpyDeviceToHost);
        hipFree(d_entries), hipFree(d_uints), hipFree(d_huff_sp);
    }

    // write metadata to fs
    io::WriteArrayToBinary(basename + ".hmeta", _counts + nchunk, 2 * nchunk);
    io::WriteArrayToBinary(basename + ".hbyte", huff_sp, total_uints);

    size_t metadata_size =
        (2 * nchunk) * sizeof(decltype(_counts)) + sizeof(Huff) * (2 * type_bitcount) + sizeof(Quant) * dict_size;

    // clean up
    hipFree(d_huff_dn), hipFree(d_sp_bits);
    delete[] huff_sp, delete[] _counts;

    return std::make_tuple(total_bits, total_uints, metadata_size);
}

/**
 * @brief experiment warpup; use after dual-quant; of anysize
 * @todo experiment only, no decoding yet
 */
template <typename Quant, typename Huff, typename Data>
void lossless::interface::HuffmanEncodeWithTree_3D(
    Index<3>::idx_t idx,
    string&         basename,
    Quant*          h_quant_in,
    size_t          len,
    int             dict_size)
{
    auto d_quant_in = mem::CreateDeviceSpaceAndMemcpyFromHost(h_quant_in, len);

    auto d_freq = mem::CreateCUDASpace<unsigned int>(dict_size);
    ::wrapper::GetFrequency(d_quant_in, len, d_freq, dict_size);
    hipFree(d_freq);
    auto h_freq = mem::CreateHostSpaceAndMemcpyFromDevice(d_freq, dict_size);

    auto entropy = GetEntropyFromFrequency(h_freq, len, dict_size);

    std::stringstream s;
    s << basename + "-" << dict_size << "-ui" << sizeof(Huff) << ".lean_cb";
    auto h_cb       = io::ReadBinaryToNewArray<Huff>(s.str(), dict_size);
    auto d_canon_cb = mem::CreateDeviceSpaceAndMemcpyFromHost(h_cb, dict_size);

    auto get_Dg = [](size_t problem_size, size_t Db) { return (problem_size + Db - 1) / Db; };

    // Huffman space in dense format (full of zeros), fix-length space
    auto d_huff_dn = mem::CreateCUDASpace<Huff>(len);
    {
        auto Db = HuffConfig::Db_encode;
        cusz::EncodeFixedLen<Quant, Huff><<<get_Dg(len, Db), Db>>>(d_quant_in, d_huff_dn, len, d_canon_cb);
        hipDeviceSynchronize();
    }

    const static int dn_chunk = 4096;
    // deflate
    auto nchunk    = (len + dn_chunk - 1) / dn_chunk;
    auto d_sp_bits = mem::CreateCUDASpace<size_t>(nchunk);
    {
        auto Db = HuffConfig::Db_deflate;
        cusz::Deflate<Huff><<<get_Dg(nchunk, Db), Db>>>(d_huff_dn, len, d_sp_bits, dn_chunk);
        hipDeviceSynchronize();
    }

    // gather metadata (without write) before gathering huff as sp on GPU
    auto   _counts    = new size_t[nchunk * 3]();
    size_t total_bits = 0, total_uints = 0;
    cusz::GatherSpHuffMetadata<Huff>(_counts, d_sp_bits, nchunk, total_bits, total_uints);

    // partially gather on GPU and copy back (TODO fully)
    auto huff_sp = new Huff[total_uints]();
    {
        auto d_huff_sp = mem::CreateCUDASpace<Huff>(total_uints);
        auto d_uints   = mem::CreateDeviceSpaceAndMemcpyFromHost(_counts, nchunk);               // sp_uints
        auto d_entries = mem::CreateDeviceSpaceAndMemcpyFromHost(_counts + nchunk * 2, nchunk);  // sp_entries
        cusz::CopyHuffmanUintsDenseToSparse<<<nchunk, 128>>>(d_huff_dn, d_huff_sp, d_entries, d_uints, dn_chunk);
        hipDeviceSynchronize();
        hipMemcpy(huff_sp, d_huff_sp, total_uints * sizeof(Huff), hipMemcpyDeviceToHost);
        hipFree(d_entries), hipFree(d_uints), hipFree(d_huff_sp);
    }

    hipFree(d_huff_dn);

    io::WriteArrayToBinary(
        basename + "_huff_" + std::to_string(len) + "_part_" + std::to_string(idx._0) + std::to_string(idx._1) +
            std::to_string(idx._2),
        huff_sp, total_uints);

    auto avg_bits = 1.0 * total_bits / len;
    auto cr       = 1.0 * len * sizeof(Data) / (total_uints * sizeof(Huff));

    LogAll(
        log_exp,                                   //
        idx._0, idx._1, idx._2, "\t",              //
        std::setprecision(4),                      //
        " \e[1mavg bitcount:", avg_bits, "\e[0m",  //
        " CR:", cr);

    delete[] huff_sp;

    hipFree(d_freq);
    hipFree(d_quant_in);
}

template <typename Quant, typename Huff, typename Data>
void lossless::interface::HuffmanDecode(
    std::string&     basename,  //
    DataPack<Quant>* quant,
    size_t           len,
    int              chunk_size,
    size_t           total_uints,
    int              dict_size)
{
    auto type_bw    = sizeof(Huff) * 8;
    auto canon_meta = sizeof(Huff) * (2 * type_bw) + sizeof(Quant) * dict_size;
    auto canon_byte = io::ReadBinaryToNewArray<uint8_t>(basename + ".canon", canon_meta);

    auto nchunk       = (len - 1) / chunk_size + 1;
    auto huff_sp      = io::ReadBinaryToNewArray<Huff>(basename + ".hbyte", total_uints);
    auto huff_sp_meta = io::ReadBinaryToNewArray<size_t>(basename + ".hmeta", 2 * nchunk);
    auto Db           = HuffConfig::Db_deflate;  // the same as deflating
    auto Dg           = (nchunk - 1) / Db + 1;

    auto d_huff_sp = mem::CreateDeviceSpaceAndMemcpyFromHost(huff_sp, total_uints);

    auto d_huff_sp_meta = mem::CreateDeviceSpaceAndMemcpyFromHost(huff_sp_meta, 2 * nchunk);
    auto d_canon_byte   = mem::CreateDeviceSpaceAndMemcpyFromHost(canon_byte, canon_meta);
    hipDeviceSynchronize();

    cusz::Decode<<<Dg, Db, canon_meta>>>(  //
        d_huff_sp, d_huff_sp_meta, quant->dptr(), len, chunk_size, nchunk, d_canon_byte, (size_t)canon_meta);
    hipDeviceSynchronize();

    // quant->template Move<transfer::d2h>(); // unnecessary if no Huffman decoding verification

    hipFree(d_huff_sp);
    hipFree(d_huff_sp_meta);
    hipFree(d_canon_byte);
    delete[] huff_sp;
    delete[] huff_sp_meta;
    delete[] canon_byte;
}

// TODO mark types using Q/H-byte binding; internally resolve UI8-UI8_2 issue
// using Q1 = QuantTrait<1>::Quant;
// using H4 = HuffTrait<4>::Huff;

// clang-format off
template std::tuple<size_t, size_t, size_t> lossless::interface::HuffmanEncode<UI1, UI4, FP4>(string&, UI1*, UI4*, uint8_t*, size_t, size_t, int, int);
template std::tuple<size_t, size_t, size_t> lossless::interface::HuffmanEncode<UI2, UI4, FP4>(string&, UI2*, UI4*, uint8_t*, size_t, size_t, int, int);
template std::tuple<size_t, size_t, size_t> lossless::interface::HuffmanEncode<UI1, UI8, FP4>(string&, UI1*, UI8*, uint8_t*, size_t, size_t, int, int);
template std::tuple<size_t, size_t, size_t> lossless::interface::HuffmanEncode<UI2, UI8, FP4>(string&, UI2*, UI8*, uint8_t*, size_t, size_t, int, int);

template void lossless::interface::HuffmanDecode<UI1, UI4, FP4>(std::string&, DataPack<UI1>*, size_t, int, size_t, int);
template void lossless::interface::HuffmanDecode<UI2, UI4, FP4>(std::string&, DataPack<UI2>*, size_t, int, size_t, int);
template void lossless::interface::HuffmanDecode<UI1, UI8, FP4>(std::string&, DataPack<UI1>*, size_t, int, size_t, int);
template void lossless::interface::HuffmanDecode<UI2, UI8, FP4>(std::string&, DataPack<UI2>*, size_t, int, size_t, int);

template void lossless::interface::HuffmanEncodeWithTree_3D<UI1, UI4>(Index<3>::idx_t, string&, UI1*, size_t, int);
template void lossless::interface::HuffmanEncodeWithTree_3D<UI1, UI8>(Index<3>::idx_t, string&, UI1*, size_t, int);
template void lossless::interface::HuffmanEncodeWithTree_3D<UI2, UI4>(Index<3>::idx_t, string&, UI2*, size_t, int);
template void lossless::interface::HuffmanEncodeWithTree_3D<UI2, UI8>(Index<3>::idx_t, string&, UI2*, size_t, int);
