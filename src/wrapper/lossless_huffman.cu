#include "hip/hip_runtime.h"
/**
 * @file lossless_huffman.cu
 * @author Jiannan Tian
 * @brief A high-level Huffman wrapper. Allocations are explicitly out of called functions.
 * @version 0.3
 * @date 2021-06-17
 *
 * (C) 2021 by Washington State University, Argonne National Laboratory
 *
 */

#include <numeric>
#include <stdexcept>
#include <type_traits>
#include "../kernel/huffman_codec.h"
#include "lossless_huffman.h"

#if __cplusplus >= 201703L
#define CONSTEXPR constexpr
#else
#define CONSTEXPR
#endif

#define H2D hipMemcpyHostToDevice
#define D2H hipMemcpyDeviceToHost

namespace {

auto get_npart = [](auto size, auto subsize) {
    static_assert(
        std::numeric_limits<decltype(size)>::is_integer and std::numeric_limits<decltype(subsize)>::is_integer,
        "[get_npart] must be plain interger types.");
    return (size + subsize - 1) / subsize;
};

template <typename T>
size_t get_nbyte(T*, size_t len)
{
    return sizeof(T) * len;
}

// TODO add __restrict__ to each
template <typename Huff, typename MetadataT = size_t>
inline void process_huffman_metadata(
    MetadataT*   d_seg_bits,     // on-device space
    MetadataT*   h_seg_bits,     // on-host space & archive
    MetadataT*   h_seg_uints,    // on-host space
    MetadataT*   h_seg_entries,  // on-host space & archive
    unsigned int nchunk,
    size_t&      num_bits,
    size_t&      num_uints)
{
    constexpr auto bitlen = sizeof(Huff) * 8;
    hipMemcpy(h_seg_bits, d_seg_bits, nchunk * sizeof(MetadataT), D2H);
    memcpy(h_seg_uints, h_seg_bits, nchunk * sizeof(MetadataT));
    for_each(h_seg_uints, h_seg_uints + nchunk, [&](MetadataT& i) { i = (i + bitlen - 1) / bitlen; });
    memcpy(h_seg_entries + 1, h_seg_uints, (nchunk - 1) * sizeof(MetadataT));
    for (auto i = 1; i < nchunk; i++) h_seg_entries[i] += h_seg_entries[i - 1];  // inclusive scan
    num_bits  = std::accumulate(h_seg_bits, h_seg_bits + nchunk, (size_t)0);
    num_uints = std::accumulate(h_seg_uints, h_seg_uints + nchunk, (size_t)0);
}

template <typename Huff, typename MetadataT = size_t>
__global__ void concatenate_huffman_segments(
    Huff* __restrict__ input_dn,
    Huff* __restrict__ output_sp,
    MetadataT* seg_entries,
    MetadataT* seg_uints,
    MetadataT  chunk_size)
{
    auto len      = seg_uints[blockIdx.x];
    auto sp_entry = seg_entries[blockIdx.x];
    auto dn_entry = chunk_size * blockIdx.x;
    auto n        = (len + blockDim.x - 1) / blockDim.x;

    for (auto i = 0; i < n; i++) {
        auto _tid = threadIdx.x + i * blockDim.x;
        if (_tid < len) *(output_sp + sp_entry + _tid) = *(input_dn + dn_entry + _tid);
        __syncthreads();
    }
}

}  // namespace

/********************************************************************************
 * high-level API
 ********************************************************************************/
#define ENC_CTX HuffmanEncodingDescriptor<Input, Huff, MetadataT>

template <typename Input, typename Huff, typename MetadataT, bool NSYMBOL_RESTRICT>
void compress_huffman_encode(ENC_CTX* ctx, Input* d_input, size_t len, int chunk_size)
{
    constexpr auto ENC_SEQ = 4;
    static_assert(
        (std::is_floating_point<Input>::value and NSYMBOL_RESTRICT) == false,
        "[compress_huffman_encode] floating-point input cannot work with symbol number restricted workflow.");
    if CONSTEXPR (NSYMBOL_RESTRICT == false) {  // TODO
        throw std::runtime_error("[compress_huffman_encode] branch(NSYMBOL_RESTRICT) not implemented.");
    }

    auto nchunk = get_npart(len, chunk_size);

    /********************************************************************************
     * encoding in a fixed-length space
     ********************************************************************************/
    {
        auto dim_block = 256;
        auto dim_grid  = get_npart(len, dim_block);
        cusz::EncodeFixedLen_cub                                       //
            <Input, Huff, ENC_SEQ><<<dim_grid, dim_block / ENC_SEQ>>>  //
            (d_input, ctx->space.fixed_len, len, ctx->space.d_book);
        hipDeviceSynchronize();
    }
    /********************************************************************************
     * deflate
     ********************************************************************************/
    {
        auto dim_block = 256;
        auto dim_grid  = get_npart(nchunk, dim_block);
        cusz::Deflate<Huff><<<dim_grid, dim_block>>>(ctx->space.fixed_len, len, ctx->space.d_seg_bits, chunk_size);
        hipDeviceSynchronize();
    }
    /********************************************************************************
     * process metadata
     ********************************************************************************/
    {
        process_huffman_metadata<Huff>(
            ctx->space.d_seg_bits, ctx->space.h_seg_bits, ctx->space.h_seg_uints, ctx->space.h_seg_entries, nchunk,
            ctx->num_bits, ctx->num_uints);

        hipMemcpy(
            ctx->space.d_seg_bits, ctx->space.h_seg_bits,  //
            get_nbyte(ctx->space.h_seg_uints, ctx->nchunk), H2D);
        hipMemcpy(
            ctx->space.d_seg_entries, ctx->space.h_seg_entries,  //
            get_nbyte(ctx->space.h_seg_entries, ctx->nchunk), H2D);
    }
    /********************************************************************************
     * concatenate segments
     ********************************************************************************/
    {
        concatenate_huffman_segments<<<nchunk, 128>>>(
            ctx->space.fixed_len, ctx->space.d_bitstream, ctx->space.d_seg_entries, ctx->space.d_seg_uints, chunk_size);
        hipDeviceSynchronize();

        hipMemcpy(
            ctx->space.h_bitstream, ctx->space.d_bitstream,  //
            get_nbyte(ctx->ctx->space.h_bitstream, ctx->num_uints), D2H);
    }
    /* EOF */
}

#define INSTANTIATE_COMPRESS_HUFFMAN_ENCODE(Input, Huff, MetadataT) \
    template <>                                                     \
    void compress_huffman_encode<Input, Huff, MetadataT>(           \
        HuffmanEncodingDescriptor<Input, Huff, MetadataT>*, Input*, size_t, int);

INSTANTIATE_COMPRESS_HUFFMAN_ENCODE(uint8_t, uint32_t, size_t)
INSTANTIATE_COMPRESS_HUFFMAN_ENCODE(uint8_t, uint32_t, unsigned int)
INSTANTIATE_COMPRESS_HUFFMAN_ENCODE(uint8_t, uint64_t, size_t)
INSTANTIATE_COMPRESS_HUFFMAN_ENCODE(uint8_t, uint64_t, unsigned int)
INSTANTIATE_COMPRESS_HUFFMAN_ENCODE(uint16_t, uint32_t, size_t)
INSTANTIATE_COMPRESS_HUFFMAN_ENCODE(uint16_t, uint32_t, unsigned int)
INSTANTIATE_COMPRESS_HUFFMAN_ENCODE(uint16_t, uint64_t, size_t)
INSTANTIATE_COMPRESS_HUFFMAN_ENCODE(uint16_t, uint64_t, unsigned int)

/****************************************************************************************************/

#define DEC_CTX HuffmanDecodingDescriptor<Output, Huff, MetadataT>

template <typename Output, typename Huff, typename MetadataT, bool NSYMBOL_RESTRICT>
void decompress_huffman_decode(DEC_CTX* ctx, Output* d_output, size_t len, int chunk_size)
{
    static_assert(
        (std::is_floating_point<Output>::value and NSYMBOL_RESTRICT) == false,
        "[compress_huffman_encode] floating-point input cannot work with symbol number restricted workflow.");
    if CONSTEXPR (NSYMBOL_RESTRICT == false) {  // TODO
        throw std::runtime_error("[compress_huffman_encode] branch(NSYMBOL_RESTRICT) not implemented.");
    }

    auto nchunk = get_npart(len, chunk_size);
    // clang-format off
    hipMemcpy(ctx->space.d_bitstream,   ctx->space.h_bitstream,   get_nbyte(ctx->space.h_bitstream,   ctx->num_uints),   H2D);
    hipMemcpy(ctx->space.d_seg_bits,    ctx->space.h_seg_bits,    get_nbyte(ctx->space.h_seg_bits,    ctx->nchunk),      H2D);
    hipMemcpy(ctx->space.d_seg_entries, ctx->space.h_seg_entries, get_nbyte(ctx->space.h_seg_entries, ctx->nchunk),      H2D);
    hipMemcpy(ctx->space.d_revbook,     ctx->space.h_revbook,     get_nbyte(ctx->space.h_revbook,     ctx->len.revbook), H2D);
    // clang-format on
    {
        auto dim_block = 256;  // the same as deflate
        auto dim_grid  = get_npart(nchunk, dim_block);
        huffman_decode_kernel<<<dim_grid, dim_block, ctx->len.revbook>>>(
            ctx->space.d_bitstream, ctx->space.d_seg_entries, ctx->space.d_seg_uints, d_output, chunk_size, nchunk,
            ctx->space.d_revbook, ctx->len.revbook);
        hipDeviceSynchronize();
    }
    /* EOF */
}

#define INSTANTIATE_DECOMPRESS_HUFFMAN_DECODE(Output, Huff, MetadataT) \
    template <>                                                        \
    void decompress_huffman_decode<Output, Huff, MetadataT>(           \
        HuffmanDecodingDescriptor<Output, Huff, MetadataT>*, Output*, size_t, int);

INSTANTIATE_DECOMPRESS_HUFFMAN_DECODE(uint8_t, uint32_t, size_t)
INSTANTIATE_DECOMPRESS_HUFFMAN_DECODE(uint8_t, uint32_t, unsigned int)
INSTANTIATE_DECOMPRESS_HUFFMAN_DECODE(uint8_t, uint64_t, size_t)
INSTANTIATE_DECOMPRESS_HUFFMAN_DECODE(uint8_t, uint64_t, unsigned int)
INSTANTIATE_DECOMPRESS_HUFFMAN_DECODE(uint16_t, uint32_t, size_t)
INSTANTIATE_DECOMPRESS_HUFFMAN_DECODE(uint16_t, uint32_t, unsigned int)
INSTANTIATE_DECOMPRESS_HUFFMAN_DECODE(uint16_t, uint64_t, size_t)
INSTANTIATE_DECOMPRESS_HUFFMAN_DECODE(uint16_t, uint64_t, unsigned int)
