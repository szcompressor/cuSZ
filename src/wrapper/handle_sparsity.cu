/**
 * @file handle_sparsity.cu
 * @author Jiannan Tian
 * @brief A high-level sparsity handling wrapper. Gather/scatter method to handle cuSZ prediction outlier.
 * @version 0.3
 * @date 2021-07-08
 * (created) 2020-09-10 (rev1) 2021-06-17 (rev2) 2021-07-08
 *
 * (C) 2021 by Washington State University, Argonne National Laboratory
 *
 */

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <cstddef>
#include <iostream>
#include "../utils/cuda_err.cuh"
#include "handle_sparsity.h"

using handle_t = hipsparseHandle_t;
using stream_t = hipStream_t;
using descr_t  = hipsparseMatDescr_t;

/********************************************************************************
 * "S" (for "single-precision") is used; can't generalize
 ********************************************************************************/
void compress_gather_CUDA10(struct OutlierDescriptor<float>* csr, float* in_outlier)
{
    handle_t handle       = nullptr;
    stream_t stream       = nullptr;
    descr_t  mat_desc     = nullptr;
    size_t   lworkInBytes = 0;
    char*    d_work       = nullptr;
    float    threshold    = 0;
    auto     m            = csr->m;
    auto     n            = csr->m;
    auto     lda          = csr->m;

    // clang-format off
    CHECK_CUDA(hipStreamCreateWithFlags   ( &stream,    hipStreamNonBlocking        )); // 1. create stream
    CHECK_CUSPARSE(hipsparseCreate          ( &handle                                  )); // 2. create handle
    CHECK_CUSPARSE(hipsparseSetStream       (  handle,    stream                       )); // 3. bind stream
    CHECK_CUSPARSE(hipsparseCreateMatDescr  ( &mat_desc                                )); // 4. create mat_desc
    CHECK_CUSPARSE(hipsparseSetMatIndexBase (  mat_desc,  HIPSPARSE_INDEX_BASE_ZERO     )); // zero based
    CHECK_CUSPARSE(hipsparseSetMatType      (  mat_desc,  HIPSPARSE_MATRIX_TYPE_GENERAL )); // type
    // clang-format on

    CHECK_CUSPARSE(hipsparseSpruneDense2csr_bufferSizeExt(  //
        handle, m, n, in_outlier, lda, &threshold, mat_desc, csr->pool.entry.values, csr->pool.entry.rowptr,
        csr->pool.entry.colidx, &lworkInBytes));

    if (nullptr != d_work) hipFree(d_work);
    CHECK_CUDA(hipMalloc((void**)&d_work, lworkInBytes));  // TODO where to release d_work?

    auto nnz = 0;

    /* step 4: compute rowptr and nnz */
    CHECK_CUSPARSE(hipsparseSpruneDense2csrNnz(  //
        handle, m, n, in_outlier, lda, &threshold, mat_desc, csr->pool.entry.rowptr, &nnz, d_work));
    CHECK_CUDA(hipDeviceSynchronize());

    csr->compress_configure_with_nnz(nnz);

    if (nnz == 0) {
        std::cout << "nnz == 0, exiting gather.\n";
        return;
    }

    /* step 5: compute col_ind and values */

    CHECK_CUSPARSE(hipsparseSpruneDense2csr(  //
        handle, m, n, in_outlier, lda, &threshold, mat_desc, csr->pool.entry.values, csr->pool.entry.rowptr,
        csr->pool.entry.colidx, d_work));
    CHECK_CUDA(hipDeviceSynchronize());

    if (handle) hipsparseDestroy(handle);
    if (stream) hipStreamDestroy(stream);
    if (mat_desc) hipsparseDestroyMatDescr(mat_desc);
}

void decompress_scatter_CUDA10(struct OutlierDescriptor<float>* csr, float* in_outlier)
{
    //     throw std::runtime_error("[decompress_scatter] not implemented");
    handle_t handle   = nullptr;
    stream_t stream   = nullptr;
    descr_t  mat_desc = nullptr;
    auto     m        = csr->m;
    auto     n        = csr->m;
    auto     lda      = csr->m;

    // clang-format off
    CHECK_CUDA(hipStreamCreateWithFlags   ( &stream,   hipStreamNonBlocking        )); // 1. create stream
    CHECK_CUSPARSE(hipsparseCreate          ( &handle                                 )); // 2. create handle
    CHECK_CUSPARSE(hipsparseSetStream       (  handle,   stream                       )); // 3. bind stream
    CHECK_CUSPARSE(hipsparseCreateMatDescr  ( &mat_desc                               )); // 4. create descr
    CHECK_CUSPARSE(hipsparseSetMatIndexBase (  mat_desc, HIPSPARSE_INDEX_BASE_ZERO     )); // zero based
    CHECK_CUSPARSE(hipsparseSetMatType      (  mat_desc, HIPSPARSE_MATRIX_TYPE_GENERAL )); // type
    // clang-format on

    CHECK_CUSPARSE(hipsparseScsr2dense(
        handle, m, n, mat_desc, csr->pool.entry.values, csr->pool.entry.rowptr, csr->pool.entry.colidx, in_outlier,
        lda));
    CHECK_CUDA(hipDeviceSynchronize());

    if (handle) hipsparseDestroy(handle);
    if (stream) hipStreamDestroy(stream);
    if (mat_desc) hipsparseDestroyMatDescr(mat_desc);
}
