#include "hip/hip_runtime.h"
/**
 * @file huffman_workflow.cu
 * @author Jiannan Tian, Cody Rivera (cjrivera1@crimson.ua.edu)
 * @brief Workflow of Huffman coding.
 * @version 0.1
 * @date 2020-10-24
 * Created on 2020-04-24
 *
 * @copyright (C) 2020 by Washington State University, The University of Alabama, Argonne National Laboratory
 * See LICENSE in top-level directory
 *
 */

#include <hip/hip_runtime.h>

#include <sys/stat.h>
#include <unistd.h>
#include <algorithm>
#include <bitset>
#include <cassert>
#include <cmath>
#include <functional>
#include <iostream>
#include <numeric>
#include <stdexcept>
#include <string>
#include <tuple>
#include <type_traits>
#include <vector>

#include "canonical.cuh"
#include "cuda_error_handling.cuh"
#include "cuda_mem.cuh"
#include "dbg_gpu_printing.cuh"
#include "format.hh"
#include "histogram.cuh"
#include "huffman.cuh"
#include "huffman_codec.cuh"
#include "huffman_workflow.cuh"
#include "par_huffman.cuh"
#include "timer.hh"
#include "types.hh"

#include "cascaded.hpp"
#include "nvcomp.hpp"

int ht_state_num;
int ht_all_nodes;
using uint8__t = uint8_t;

template <typename Q>
void wrapper::GetFrequency(Q* d_bcode, size_t len, unsigned int* d_freq, int dict_size)
{
    // Parameters for thread and block count optimization

    // Initialize to device-specific values
    int deviceId;
    int maxbytes;
    int maxbytesOptIn;
    int numSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&maxbytes, hipDeviceAttributeMaxSharedMemoryPerBlock, deviceId);
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    // Account for opt-in extra shared memory on certain architectures
    hipDeviceGetAttribute(&maxbytesOptIn, hipDeviceAttributeSharedMemPerBlockOptin, deviceId);
    maxbytes = std::max(maxbytes, maxbytesOptIn);

    // Optimize launch
    int numBuckets     = dict_size;
    int numValues      = len;
    int itemsPerThread = 1;
    int RPerBlock      = (maxbytes / (int)sizeof(int)) / (numBuckets + 1);
    int numBlocks      = numSMs;
    hipFuncSetAttribute(reinterpret_cast<const void*>(p2013Histogram<Q), unsigned int>, hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
    // fits to size
    int threadsPerBlock = ((((numValues / (numBlocks * itemsPerThread)) + 1) / 64) + 1) * 64;
    while (threadsPerBlock > 1024) {
        if (RPerBlock <= 1) { threadsPerBlock = 1024; }
        else {
            RPerBlock /= 2;
            numBlocks *= 2;
            threadsPerBlock = ((((numValues / (numBlocks * itemsPerThread)) + 1) / 64) + 1) * 64;
        }
    }
    p2013Histogram                                                                      //
        <<<numBlocks, threadsPerBlock, ((numBuckets + 1) * RPerBlock) * sizeof(int)>>>  //
        (d_bcode, d_freq, numValues, numBuckets, RPerBlock);
    hipDeviceSynchronize();

    // TODO make entropy optional
    {
        auto   freq    = mem::CreateHostSpaceAndMemcpyFromDevice(d_freq, dict_size);
        double entropy = 0.0;
        for (auto i = 0; i < dict_size; i++)
            if (freq[i]) {
                auto possibility = freq[i] / (1.0 * len);
                entropy -= possibility * log(possibility);
            }
        logall(log_dbg, "entropy:", entropy);
        delete[] freq;
    }

#ifdef DEBUG_PRINT
    print_histogram<unsigned int><<<1, 32>>>(d_freq, dict_size, dict_size / 2);
    hipDeviceSynchronize();
#endif
}

template <typename H>
void PrintChunkHuffmanCoding(
    size_t* dH_bit_meta,  //
    size_t* dH_uInt_meta,
    size_t  len,
    int     chunk_size,
    size_t  total_bits,
    size_t  total_uInts)
{
    cout << "\n" << log_dbg << "Huffman coding detail start ------" << endl;
    printf("| %s\t%s\t%s\t%s\t%9s\n", "chunk", "bits", "bytes", "uInt", "chunkCR");
    for (size_t i = 0; i < 8; i++) {
        size_t n_byte   = (dH_bit_meta[i] - 1) / 8 + 1;
        auto   chunk_CR = ((double)chunk_size * sizeof(float) / (1.0 * (double)dH_uInt_meta[i] * sizeof(H)));
        printf("| %lu\t%lu\t%lu\t%lu\t%9.6lf\n", i, dH_bit_meta[i], n_byte, dH_uInt_meta[i], chunk_CR);
    }
    cout << "| ..." << endl
         << "| Huff.total.bits:\t" << total_bits << endl
         << "| Huff.total.bytes:\t" << total_uInts * sizeof(H) << endl
         << "| Huff.CR (uInt):\t" << (double)len * sizeof(float) / (total_uInts * 1.0 * sizeof(H)) << endl;
    cout << log_dbg << "coding detail end ----------------" << endl;
    cout << endl;
}

template <typename Q, typename H, typename DATA>
std::tuple<size_t, size_t, size_t, bool> HuffmanEncode(string& f_in, Q* d_in, size_t len, int chunk_size, bool gzip_in_use, int dict_size)
{
    // histogram
    ht_state_num = 2 * dict_size;
    ht_all_nodes = 2 * ht_state_num;
    auto d_freq  = mem::CreateCUDASpace<unsigned int>(ht_all_nodes);
    wrapper::GetFrequency(d_in, len, d_freq, dict_size);

    // Allocate cb memory
    auto d_canonical_cb = mem::CreateCUDASpace<H>(dict_size, 0xff);
    // canonical Huffman; follows H to decide first and entry type
    auto type_bw = sizeof(H) * 8;
    // first, entry, reversed codebook
    // CHANGED first and entry to H type
    auto decode_meta_size = sizeof(H) * (2 * type_bw) + sizeof(Q) * dict_size;
    auto d_decode_meta    = mem::CreateCUDASpace<uint8_t>(decode_meta_size);

    // Get codebooks
    ParGetCodebook<Q, H>(dict_size, d_freq, d_canonical_cb, d_decode_meta);
    hipDeviceSynchronize();

    auto decode_meta = mem::CreateHostSpaceAndMemcpyFromDevice(d_decode_meta, decode_meta_size);

    // Non-deflated output
    auto d_h = mem::CreateCUDASpace<H>(len);

    // --------------------------------
    // this is for internal evaluation, not in sz archive
    // auto cb_dump = mem::CreateHostSpaceAndMemcpyFromDevice(d_canonical_cb, dict_size);
    // io::WriteBinaryFile(cb_dump, dict_size, new string(f_in + ".canonized"));
    // --------------------------------

    // fix-length space
    {
        auto blockDim = tBLK_ENCODE;
        auto gridDim  = (len - 1) / blockDim + 1;
        EncodeFixedLen<Q, H><<<gridDim, blockDim>>>(d_in, d_h, len, d_canonical_cb);
        hipDeviceSynchronize();
    }

    // deflate
    auto n_chunk       = (len - 1) / chunk_size + 1;  // |
    auto d_h_bitwidths = mem::CreateCUDASpace<size_t>(n_chunk);
    // cout << log_dbg << "Huff.chunk x #:\t" << chunk_size << " x " << n_chunk << endl;
    {
        auto blockDim = tBLK_DEFLATE;
        auto gridDim  = (n_chunk - 1) / blockDim + 1;
        Deflate<H><<<gridDim, blockDim>>>(d_h, len, d_h_bitwidths, chunk_size);
        hipDeviceSynchronize();
    }

    // dump TODO change to int
    auto h_meta        = new size_t[n_chunk * 3]();
    auto dH_uInt_meta  = h_meta;
    auto dH_bit_meta   = h_meta + n_chunk;
    auto dH_uInt_entry = h_meta + n_chunk * 2;
    // copy back densely Huffman code (dHcode)
    hipMemcpy(dH_bit_meta, d_h_bitwidths, n_chunk * sizeof(size_t), hipMemcpyDeviceToHost);
    // transform in uInt
    memcpy(dH_uInt_meta, dH_bit_meta, n_chunk * sizeof(size_t));
    for_each(dH_uInt_meta, dH_uInt_meta + n_chunk, [&](size_t& i) { i = (i - 1) / (sizeof(H) * 8) + 1; });
    // make it entries
    memcpy(dH_uInt_entry + 1, dH_uInt_meta, (n_chunk - 1) * sizeof(size_t));
    for (auto i = 1; i < n_chunk; i++) dH_uInt_entry[i] += dH_uInt_entry[i - 1];

    // sum bits from each chunk
    auto total_bits  = std::accumulate(dH_bit_meta, dH_bit_meta + n_chunk, (size_t)0);
    auto total_uInts = std::accumulate(dH_uInt_meta, dH_uInt_meta + n_chunk, (size_t)0);

    auto fmt_enc1 = "Huffman enc: (#) " + std::to_string(n_chunk) + " x " + std::to_string(chunk_size);
    auto fmt_enc2 = std::to_string(total_uInts) + " " + std::to_string(sizeof(H)) + "-byte words or " +
                    std::to_string(total_bits) + " bits";
    logall(log_dbg, fmt_enc1, "=>", fmt_enc2);

    // print densely metadata
    // PrintChunkHuffmanCoding<H>(dH_bit_meta, dH_uInt_meta, len, chunk_size, total_bits, total_uInts);

    // copy back densely Huffman code in units of uInt (regarding endianness)
    // TODO reinterpret_cast
    auto h = new H[total_uInts]();
    for (auto i = 0; i < n_chunk; i++) {
        hipMemcpy(
            h + dH_uInt_entry[i],         // dst
            d_h + i * chunk_size,         // src
            dH_uInt_meta[i] * sizeof(H),  // len in H-uint
            hipMemcpyDeviceToHost);
    }

    bool nvcomp_in_use=false;
    if(!gzip_in_use && len*4/sizeof(H)/total_uInts>=30){
        int* uncompressed_data;
        const size_t in_bytes = sizeof(H) * total_uInts;
        hipMalloc(&uncompressed_data, in_bytes);
        hipMemcpy(uncompressed_data, h, in_bytes, hipMemcpyHostToDevice);
        hipStream_t stream;
        hipStreamCreate(&stream);
        //2 layers RLE, 1 Delta encoding, bitpacking enabled
        nvcomp::CascadedCompressor<int> compressor(uncompressed_data, in_bytes/sizeof(int), 2, 1, true);
        const size_t temp_size = compressor.get_temp_size();
        void * temp_space;
        hipMalloc(&temp_space, temp_size);
        size_t output_size = compressor.get_max_output_size(temp_space, temp_size);
        void * output_space;
        hipMalloc(&output_space, output_size);
        compressor.compress_async(temp_space, temp_size, output_space, &output_size, stream);
        hipStreamSynchronize(stream);

        delete[] h;
        total_uInts=output_size/sizeof(H);
        h=new H[total_uInts]();
        hipMemcpy(h, output_space, output_size, hipMemcpyDeviceToHost);
        hipFree(uncompressed_data);
        hipFree(temp_space);
        hipFree(output_space);
        hipStreamDestroy(stream);

        //record nvcomp status in metadata
        nvcomp_in_use=true;
    }

    auto time_a = hires::now();
    // dump bit_meta and uInt_meta
    io::WriteArrayToBinary(f_in + ".hmeta", h_meta + n_chunk, (2 * n_chunk));
    // write densely Huffman code and its metadata
    io::WriteArrayToBinary(f_in + ".hbyte", h, total_uInts);
    // to save first, entry and keys
    io::WriteArrayToBinary(
        f_in + ".canon",                                   //
        reinterpret_cast<uint8_t*>(decode_meta),           //
        sizeof(H) * (2 * type_bw) + sizeof(Q) * dict_size  // first, entry, reversed dict (keys)
    );
    auto time_z = hires::now();
    logall(log_dbg, "time writing Huff. binary:", static_cast<duration_t>(time_z - time_a).count(), "sec");

    size_t metadata_size = (2 * n_chunk) * sizeof(decltype(h_meta))              //
                           + sizeof(H) * (2 * type_bw) + sizeof(Q) * dict_size;  // uint8_t

    //////// clean up
    hipFree(d_freq);
    hipFree(d_canonical_cb);
    hipFree(d_decode_meta);
    hipFree(d_h);
    hipFree(d_h_bitwidths);
    delete[] h;
    delete[] h_meta;
    delete[] decode_meta;

    return std::make_tuple(total_bits, total_uInts, metadata_size, nvcomp_in_use);
}

template <typename Q, typename H, typename DATA>
Q* HuffmanDecode(
    std::string& f_bcode_base,  //
    size_t       len,
    int          chunk_size,
    int          total_uInts,
    bool         nvcomp_in_use,
    int          dict_size)
{
    auto type_bw             = sizeof(H) * 8;
    auto canonical_meta      = sizeof(H) * (2 * type_bw) + sizeof(Q) * dict_size;
    auto canonical_singleton = io::ReadBinaryFile<uint8_t>(f_bcode_base + ".canon", canonical_meta);
    hipDeviceSynchronize();

    auto n_chunk  = (len - 1) / chunk_size + 1;
    auto hcode    = io::ReadBinaryFile<H>(f_bcode_base + ".hbyte", total_uInts);

    auto dH_meta  = io::ReadBinaryFile<size_t>(f_bcode_base + ".hmeta", 2 * n_chunk);
    auto blockDim = tBLK_DEFLATE;  // the same as deflating
    auto gridDim  = (n_chunk - 1) / blockDim + 1;

    auto d_xbcode              = mem::CreateCUDASpace<Q>(len);
    auto d_dHcode              = mem::CreateDeviceSpaceAndMemcpyFromHost(hcode, total_uInts);

    //if nvcomp is used to compress *.hbyte
    if(nvcomp_in_use){
        hipStream_t stream;
        hipStreamCreate(&stream);

        nvcomp::Decompressor<int> decompressor(d_dHcode, total_uInts*sizeof(H), stream);
        const size_t temp_size = decompressor.get_temp_size();
        void * temp_space;
        hipMalloc(&temp_space, temp_size);

        const size_t output_count = decompressor.get_num_elements();
        int * output_space;
        hipMalloc((void**)&output_space, output_count*sizeof(int));

        decompressor.decompress_async(temp_space, temp_size, output_space, output_count, stream);

        hipStreamSynchronize(stream);
        hipFree(d_dHcode);
        
        d_dHcode=mem::CreateCUDASpace<H>((unsigned long)(output_count*sizeof(int)));
        hipMemcpy(d_dHcode,output_space,output_count*sizeof(int),hipMemcpyDeviceToDevice);
        total_uInts=output_count*sizeof(int)/sizeof(H);

        hipFree(output_space);

        hipStreamDestroy(stream);
        hipFree(temp_space);
    }

    auto d_hcode_meta          = mem::CreateDeviceSpaceAndMemcpyFromHost(dH_meta, 2 * n_chunk);
    auto d_canonical_singleton = mem::CreateDeviceSpaceAndMemcpyFromHost(canonical_singleton, canonical_meta);
    hipDeviceSynchronize();

    Decode<<<gridDim, blockDim, canonical_meta>>>(  //
        d_dHcode, d_hcode_meta, d_xbcode, len, chunk_size, n_chunk, d_canonical_singleton, (size_t)canonical_meta);
    hipDeviceSynchronize();

    auto xbcode = mem::CreateHostSpaceAndMemcpyFromDevice(d_xbcode, len);
    hipFree(d_xbcode);
    hipFree(d_dHcode);
    hipFree(d_hcode_meta);
    hipFree(d_canonical_singleton);
    delete[] hcode;
    delete[] dH_meta;
    delete[] canonical_singleton;

    return xbcode;
}

template void wrapper::GetFrequency<uint8__t>(uint8__t*, size_t, unsigned int*, int);
template void wrapper::GetFrequency<uint16_t>(uint16_t*, size_t, unsigned int*, int);
template void wrapper::GetFrequency<uint32_t>(uint32_t*, size_t, unsigned int*, int);

template void PrintChunkHuffmanCoding<uint32_t>(size_t*, size_t*, size_t, int, size_t, size_t);
template void PrintChunkHuffmanCoding<uint64_t>(size_t*, size_t*, size_t, int, size_t, size_t);

template tuple3ul HuffmanEncode<uint8__t, uint32_t, float>(string&, uint8__t*, size_t, int, bool, int);
template tuple3ul HuffmanEncode<uint16_t, uint32_t, float>(string&, uint16_t*, size_t, int, bool, int);
template tuple3ul HuffmanEncode<uint32_t, uint32_t, float>(string&, uint32_t*, size_t, int, bool, int);
template tuple3ul HuffmanEncode<uint8__t, uint64_t, float>(string&, uint8__t*, size_t, int, bool, int);
template tuple3ul HuffmanEncode<uint16_t, uint64_t, float>(string&, uint16_t*, size_t, int, bool, int);
template tuple3ul HuffmanEncode<uint32_t, uint64_t, float>(string&, uint32_t*, size_t, int, bool, int);

template uint8__t* HuffmanDecode<uint8__t, uint32_t, float>(std::string&, size_t, int, int, bool, int);
template uint16_t* HuffmanDecode<uint16_t, uint32_t, float>(std::string&, size_t, int, int, bool, int);
template uint32_t* HuffmanDecode<uint32_t, uint32_t, float>(std::string&, size_t, int, int, bool, int);
template uint8__t* HuffmanDecode<uint8__t, uint64_t, float>(std::string&, size_t, int, int, bool, int);
template uint16_t* HuffmanDecode<uint16_t, uint64_t, float>(std::string&, size_t, int, int, bool, int);
template uint32_t* HuffmanDecode<uint32_t, uint64_t, float>(std::string&, size_t, int, int, bool, int);
// clang-format off
