/**
 * @file cusz.cu
 * @author Jiannan Tian
 * @brief Driver program of cuSZ.
 * @version 0.1
 * @date 2020-09-20
 * Created on 2019-12-30
 *
 * @copyright (C) 2020 by Washington State University, The University of Alabama, Argonne National Laboratory
 * See LICENSE in top-level directory
 *
 */

#include <math.h>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>
#include <cmath>
#include <cstddef>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <string>
#include <unordered_map>
#include <vector>

using std::string;

#if __cplusplus >= 201103L

#include "analysis_utils.hh"
#include "argparse.hh"
#include "cusz_interface.cuh"
#include "filter.cuh"
#include "gtest/gtest.h"
#include "metadata.hh"
#include "pack.hh"
#include "query.hh"
#include "type_aliasing.hh"
#include "types.hh"
#include "utils/cuda_err.cuh"
#include "utils/cuda_mem.cuh"
#include "utils/format.hh"
#include "utils/io.hh"
#include "utils/timer.hh"

double expectedErr;
double actualAbsErr;
double actualRelErr;
string z_mode;

void InitializeDims(argpack* ap)
{
    std::unordered_map<std::string, std::vector<int>> dataset_entries = {
        {std::string("hacc"), {280953867, 1, 1, 1, 1}},    {std::string("hacc1b"), {1073726487, 1, 1, 1, 1}},
        {std::string("cesm"), {3600, 1800, 1, 1, 2}},      {std::string("hurricane"), {500, 500, 100, 1, 3}},
        {std::string("nyx-s"), {512, 512, 512, 1, 3}},     {std::string("nyx-m"), {1024, 1024, 1024, 1, 3}},
        {std::string("qmc"), {288, 69, 7935, 1, 3}},       {std::string("qmcpre"), {69, 69, 33120, 1, 3}},
        {std::string("exafel"), {388, 59200, 1, 1, 2}},    {std::string("aramco"), {235, 849, 849, 1, 3}},
        {std::string("parihaka"), {1168, 1126, 922, 1, 3}}};

    if (not ap->demo_dataset.empty()) {
        // TODO try-catch
        auto dim4_datum = dataset_entries.at(ap->demo_dataset);

        ap->dim4._0 = (int)dim4_datum[0];
        ap->dim4._1 = (int)dim4_datum[1];
        ap->dim4._2 = (int)dim4_datum[2];
        ap->dim4._3 = (int)dim4_datum[3];
        ap->ndim    = (int)dim4_datum[4];
    }

    if (ap->ndim == 1)
        ap->GPU_block_size = MetadataTrait<1>::Block;
    else if (ap->ndim == 2)
        ap->GPU_block_size = MetadataTrait<2>::Block;
    else if (ap->ndim == 3)
        ap->GPU_block_size = MetadataTrait<3>::Block;

    auto get_nblk = [&](int d) { return (d + ap->GPU_block_size - 1) / ap->GPU_block_size; };

    ap->nblk4._0 = get_nblk(ap->dim4._0);
    ap->nblk4._1 = get_nblk(ap->dim4._1);
    ap->nblk4._2 = get_nblk(ap->dim4._2);
    ap->nblk4._3 = get_nblk(ap->dim4._3);

    ap->len = ap->dim4._0 * ap->dim4._1 * ap->dim4._2 * ap->dim4._3;

    ap->stride4 = {
        1,                          //
        ap->dim4._0,                //
        ap->dim4._0 * ap->dim4._1,  //
        ap->dim4._0 * ap->dim4._1 * ap->dim4._2};
}

void CheckShellCall(string cmd_string)
{
    char* cmd = new char[cmd_string.length() + 1];
    strcpy(cmd, cmd_string.c_str());
    int status = system(cmd);
    delete[] cmd;
    cmd = nullptr;
    if (status < 0) { LogAll(log_err, "Shell command call failed, exit code: ", errno, "->", strerror(errno)); }
}

TEST(cuSZTest, TestMaxError)
{
    double actualErr = (z_mode == "r2r") ? actualRelErr : actualAbsErr;
    ASSERT_LE(actualErr, expectedErr);
}

template <typename Data, int DownscaleFactor, int tBLK>
Data* pre_binning(Data* d, size_t* dim_array)
{
    return nullptr;
}

int main(int argc, char** argv)
{
    auto ap = new ArgPack();
    ap->ParseCuszArgs(argc, argv);

    int    nnz_outlier = 0;
    size_t total_bits, total_uInt, huff_meta_size;
    bool   nvcomp_in_use = false;

    if (ap->verbose) {
        GetMachineProperties();
        GetDeviceProperty();
    }

    auto& wf       = ap->szwf;
    auto& subfiles = ap->subfiles;

    // TODO hardcode for float for now
    using DataInUse                  = float;
    struct DataPack<DataInUse>* adp  = nullptr;
    DataInUse*                  data = nullptr;

    if (wf.lossy_construct or wf.lossy_dryrun) {
        InitializeDims(ap);

        LogAll(
            log_info, "load", subfiles.cx_path2file, ap->len * (ap->dtype == "f32" ? sizeof(float) : sizeof(double)),
            "bytes,", ap->dtype);

        auto len = ap->len;

        auto m   = cusz::impl::GetEdgeOfReinterpretedSquare(len);  // row-major mxn matrix
        auto mxm = m * m;

        LogAll(log_dbg, "add padding:", m, "units");

        auto a = hires::now();
        CHECK_CUDA(hipHostMalloc(&data, mxm * sizeof(DataInUse)));
        memset(data, 0x00, mxm * sizeof(DataInUse));
        io::ReadBinaryToArray<DataInUse>(subfiles.cx_path2file, data, len);
        DataInUse* d_data = mem::CreateDeviceSpaceAndMemcpyFromHost(data, mxm);
        auto       z      = hires::now();

        LogAll(log_dbg, "time loading datum:", static_cast<duration_t>(z - a).count(), "sec");

        adp = new DataPack<DataInUse>(data, d_data, len);

        if (ap->mode == "r2r") {
            double rng;
            auto   time_0 = hires::now();
            // TODO move to data analytics
            // ------------------------------------------------------------
            thrust::device_ptr<float> g_ptr = thrust::device_pointer_cast(d_data);

            size_t min_el_loc = thrust::min_element(g_ptr, g_ptr + len) - g_ptr;  // excluding padded
            size_t max_el_loc = thrust::max_element(g_ptr, g_ptr + len) - g_ptr;  // excluding padded

            double min_value = *(g_ptr + min_el_loc);
            double max_value = *(g_ptr + max_el_loc);
            rng              = max_value - min_value;
            // ------------------------------------------------------------
            auto time_1 = hires::now();

            LogAll(log_dbg, "time scanning:", static_cast<duration_t>(time_1 - time_0).count(), "sec");

            ap->eb *= rng;
        }

        LogAll(
            log_dbg, std::to_string(ap->quant_byte) + "-byte quant type,",
            std::to_string(ap->huff_byte) + "-byte internal Huff type");
    }

    if (wf.pre_binning) {
        cerr << log_err
             << "Binning is not working temporarily; we are improving end-to-end throughput by NOT touching "
                "filesystem. (ver. 0.1.4)"
             << endl;
        exit(1);
    }

    if (wf.lossy_construct or wf.lossy_dryrun) {  // fp32 only for now

        if (ap->quant_byte == 1) {
            if (ap->huff_byte == 4)
                cusz::interface::Compress<true, 4, 1, 4>(
                    ap, adp, nnz_outlier, total_bits, total_uInt, huff_meta_size, nvcomp_in_use);
            else
                cusz::interface::Compress<true, 4, 1, 8>(
                    ap, adp, nnz_outlier, total_bits, total_uInt, huff_meta_size, nvcomp_in_use);
        }
        else if (ap->quant_byte == 2) {
            if (ap->huff_byte == 4)
                cusz::interface::Compress<true, 4, 2, 4>(
                    ap, adp, nnz_outlier, total_bits, total_uInt, huff_meta_size, nvcomp_in_use);
            else
                cusz::interface::Compress<true, 4, 2, 8>(
                    ap, adp, nnz_outlier, total_bits, total_uInt, huff_meta_size, nvcomp_in_use);
        }

        // pack metadata
        auto mp = new metadata_pack();
        PackMetadata(ap, mp, nnz_outlier);
        mp->total_bits     = total_bits;
        mp->total_uInt     = total_uInt;
        mp->huff_meta_size = huff_meta_size;
        mp->nvcomp_in_use  = nvcomp_in_use;

        auto mp_byte = reinterpret_cast<char*>(mp);
        // yet another metadata package
        io::WriteArrayToBinary(subfiles.c_fo_yamp, mp_byte, sizeof(metadata_pack));

        delete mp;
    }

    if (data and adp) {
        hipHostFree(data);  // really messy considering adp pointers are freed elsewhere
        data = nullptr;
        delete adp;
    }

    // invoke system() to untar archived files first before decompression
    if (not wf.lossy_construct and wf.lossy_reconstruct) {
        string cx_directory = subfiles.cx_path2file.substr(0, subfiles.cx_path2file.rfind('/') + 1);
        string cmd_string;
        if (cx_directory.length() == 0)
            cmd_string = "tar -xf " + subfiles.cx_path2file + ".sz";
        else
            cmd_string = "tar -xf " + subfiles.cx_path2file + ".sz" + " -C " + cx_directory;

        CheckShellCall(cmd_string);
    }

    if (wf.lossy_reconstruct) {  // fp32 only for now

        // unpack metadata
        auto mp_byte = io::ReadBinaryToNewArray<char>(subfiles.x_fi_yamp, sizeof(metadata_pack));
        auto mp      = reinterpret_cast<metadata_pack*>(mp_byte);

        UnpackMetadata(ap, mp, nnz_outlier);
        total_bits     = mp->total_bits;
        total_uInt     = mp->total_uInt;
        huff_meta_size = mp->huff_meta_size;
        nvcomp_in_use  = mp->nvcomp_in_use;

        if (ap->quant_byte == 1) {
            if (ap->huff_byte == 4)
                cusz::interface::Decompress<true, 4, 1, 4>(
                    ap, nnz_outlier, total_bits, total_uInt, huff_meta_size, nvcomp_in_use);
            else if (ap->huff_byte == 8)
                cusz::interface::Decompress<true, 4, 1, 8>(
                    ap, nnz_outlier, total_bits, total_uInt, huff_meta_size, nvcomp_in_use);
        }
        else if (ap->quant_byte == 2) {
            if (ap->huff_byte == 4)
                cusz::interface::Decompress<true, 4, 2, 4>(
                    ap, nnz_outlier, total_bits, total_uInt, huff_meta_size, nvcomp_in_use);
            else if (ap->huff_byte == 8)
                cusz::interface::Decompress<true, 4, 2, 8>(
                    ap, nnz_outlier, total_bits, total_uInt, huff_meta_size, nvcomp_in_use);
        }
    }

    // invoke system() function to merge and compress the resulting 5 files after cusz compression
    string cx_basename = subfiles.cx_path2file.substr(subfiles.cx_path2file.rfind('/') + 1);
    if (not wf.lossy_reconstruct and wf.lossy_construct) {
        auto tar_a = hires::now();

        // remove *.sz if existing
        string cmd_string = "rm -rf " + ap->opath + cx_basename + ".sz";
        CheckShellCall(cmd_string);

        // using tar command to encapsulate files
        string files_for_merging;
        if (wf.skip_huffman_enc) {
            files_for_merging = cx_basename + ".outlier " + cx_basename + ".quant " + cx_basename + ".yamp";
        }
        else {
            files_for_merging = cx_basename + ".hbyte " + cx_basename + ".outlier " + cx_basename + ".canon " +
                                cx_basename + ".hmeta " + cx_basename + ".yamp";
        }
        if (wf.lossless_gzip) {
            cmd_string = "cd " + ap->opath + ";tar -czf " + cx_basename + ".sz " + files_for_merging;
        }
        else {
            cmd_string = "cd " + ap->opath + ";tar -cf " + cx_basename + ".sz " + files_for_merging;
        }
        CheckShellCall(cmd_string);

        // remove 5 subfiles
        cmd_string = "cd " + ap->opath + ";rm -rf " + files_for_merging;
        CheckShellCall(cmd_string);

        auto tar_z = hires::now();

        auto ad_hoc_fix = ap->opath.substr(0, ap->opath.size() - 1);
        LogAll(log_dbg, "time tar'ing:", static_cast<duration_t>(tar_z - tar_a).count(), "sec");
        LogAll(log_info, "output:", ad_hoc_fix + cx_basename + ".sz");
    }

    // if it's decompression, remove released subfiles at last.
    if (not wf.lossy_construct and wf.lossy_reconstruct) {
        string files_for_deleting;
        if (wf.skip_huffman_enc) {
            files_for_deleting = cx_basename + ".outlier " + cx_basename + ".quant " + cx_basename + ".yamp";
        }
        else {
            files_for_deleting = cx_basename + ".hbyte " + cx_basename + ".outlier " + cx_basename + ".canon " +
                                 cx_basename + ".hmeta " + cx_basename + ".yamp";
        }
        string cmd_string =
            "cd " + subfiles.cx_path2file.substr(0, subfiles.cx_path2file.rfind('/')) + ";rm -rf " + files_for_deleting;
        CheckShellCall(cmd_string);
    }

    if (wf.lossy_construct and wf.lossy_reconstruct) {
        // remove *.sz if existing
        string cmd_string = "rm -rf " + ap->opath + cx_basename + ".sz";
        CheckShellCall(cmd_string);

        // using tar command to encapsulate files
        string files_for_merging;
        if (wf.skip_huffman_enc) {
            files_for_merging = cx_basename + ".outlier " + cx_basename + ".quant " + cx_basename + ".yamp";
        }
        else {
            files_for_merging = cx_basename + ".hbyte " + cx_basename + ".outlier " + cx_basename + ".canon " +
                                cx_basename + ".hmeta " + cx_basename + ".yamp";
        }
        if (wf.lossless_gzip) {
            cmd_string = "cd " + ap->opath + ";tar -czf " + cx_basename + ".sz " + files_for_merging;
        }
        else {
            cmd_string = "cd " + ap->opath + ";tar -cf " + cx_basename + ".sz " + files_for_merging;
        }
        CheckShellCall(cmd_string);

        // remove 5 subfiles
        cmd_string = "cd " + ap->opath + ";rm -rf " + files_for_merging;
        CheckShellCall(cmd_string);

        LogAll(log_info, "write to: " + ap->opath + cx_basename + ".sz");
        LogAll(log_info, "write to: " + ap->opath + cx_basename + ".szx");

        if (wf.gtest) {
            expectedErr  = ap->eb;
            z_mode       = ap->mode;
            auto stat    = ap->stat;
            actualAbsErr = stat.max_abserr;
            actualRelErr = stat.max_abserr_vs_range;
            ::testing::InitGoogleTest(&argc, argv);
            return RUN_ALL_TESTS();
        }
    }
}

#endif
