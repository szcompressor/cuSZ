/**
 * @file cusz_workflow.cu
 * @author Jiannan Tian
 * @brief Workflow of cuSZ.
 * @version 0.2
 * @date 2021-01-16
 * (create) 2020-02-12; (release) 2020-09-20; (rev1) 2021-01-16
 *
 * @copyright (C) 2020 by Washington State University, The University of Alabama, Argonne National Laboratory
 * See LICENSE in top-level directory
 *
 */

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include <cxxabi.h>
#include <bitset>
#include <cstdlib>
#include <exception>
#include <iostream>
#include <typeinfo>

// #if __cplusplus >= 201103L

#include <type_traits>

//#include "analysis_utils.hh"
#include "argparse.hh"
#include "autotune.hh"
#include "cusz_interface.cuh"
#include "dryrun.cuh"
#include "dualquant.cuh"
#include "gather_scatter.cuh"
#include "huff_interface.cuh"
#include "lorenzo_trait.cuh"
#include "metadata.hh"
#include "type_trait.hh"
#include "utils/cuda_err.cuh"
#include "utils/cuda_mem.cuh"
#include "utils/format.hh"
#include "utils/io.hh"
#include "utils/verify.hh"

using std::cerr;
using std::cout;
using std::endl;
using std::string;

namespace fm = cusz::predictor_quantizer;
namespace dr = cusz::dryrun;

/*
template <typename Data, typename Quant>
void cusz::impl::VerifyHuffman(
    string const& fi,
    size_t        len,
    Quant*        xq,
    int           chunk_size,
    size_t*       dims,
    double*       eb_variants)
{
    LogAll(log_info, "Redo PdQ just to get quant data.");

    auto  veri_data   = io::ReadBinaryToNewArray<Data>(fi, len);
    Data* veri_d_data = mem::CreateDeviceSpaceAndMemcpyFromHost(veri_data, len);
    auto  veri_d_q    = mem::CreateCUDASpace<Quant>(len);

    PdQ(veri_d_data, veri_d_q, dims, eb_variants);

    auto veri_q = mem::CreateHostSpaceAndMemcpyFromDevice(veri_d_q, len);

    auto count = 0;
    for (auto i = 0; i < len; i++)
        if (xq[i] != veri_q[i]) count++;
    if (count != 0)
        LogAll(log_err, "percentage of not being equal:", count / (1.0 * len));
    else
        LogAll(log_info, "Decoded correctly.");

    if (count != 0) {
        auto n_chunk = (len - 1) / chunk_size + 1;
        for (auto c = 0; c < n_chunk; c++) {
            auto chunk_id_printed = false, prev_point_printed = false;
            for (auto i = 0; i < chunk_size; i++) {
                auto idx = i + c * chunk_size;
                if (idx >= len) break;
                if (xq[idx] != xq[idx]) {
                    if (not chunk_id_printed) {
                        cerr << "chunk id: " << c << "\t"
                             << "start@ " << c * chunk_size << "\tend@ " << (c + 1) * chunk_size - 1 << endl;
                        chunk_id_printed = true;
                    }
                    if (not prev_point_printed) {
                        if (idx != c * chunk_size)  // not first point
                            cerr << "PREV-idx:" << idx - 1 << "\t" << xq[idx - 1] << "\t" << xq[idx - 1] << endl;
                        else
                            cerr << "wrong at first point!" << endl;
                        prev_point_printed = true;
                    }
                    cerr << "idx:" << idx << "\tdecoded: " << xq[idx] << "\tori: " << xq[idx] << endl;
                }
            }
        }
    }

    hipFree(veri_d_q);
    hipFree(veri_d_data);
    delete[] veri_q;
    delete[] veri_data;
}
 */

template <typename T>
auto CopyToBuffer_3D(
    T* __restrict buffer_dst,
    T* __restrict origin_src,
    size_t          portal,
    Index<3>::idx_t part_dims,
    Index<3>::idx_t block_stride,
    Index<3>::idx_t global_stride)
{
    for (auto k = 0; k < part_dims._2; k++)
        for (auto j = 0; j < part_dims._1; j++)
            for (auto i = 0; i < part_dims._0; i++)
                buffer_dst[i + j * block_stride._1 + k * block_stride._2] =
                    origin_src[portal + (i + j * global_stride._1 + k * global_stride._2)];
}

template <typename T, int N = 3>
auto PrintBuffer(T* data, size_t start, Integer3 strides)
{
    cout << "printing buffer\n";
    for (auto k = 0; k < N; k++) {
        for (auto j = 0; j < N; j++) {
            for (auto i = 0; i < N; i++) {  //
                cout << data[start + (i + j * strides._1 + k * strides._2)] << " ";
            }
            cout << "\n";
        }
    }
    cout << endl;
};

// clang-format off
template <bool If_FP, int DataByte, int QuantByte, int HuffByte>
void cusz::interface::Compress(
    argpack* ap,
    struct DataPack<typename DataTrait<If_FP, DataByte>::Data>* adp,
    int&     nnz_outlier,
    size_t&  num_bits,
    size_t&  num_uints,
    size_t&  huff_meta_size,
    bool&    nvcomp_in_use)
{
    // clang-format on
    using Data  = typename DataTrait<If_FP, DataByte>::Data;
    using Quant = typename QuantTrait<QuantByte>::Quant;
    using Huff  = typename HuffTrait<HuffByte>::Huff;

    size_t len = ap->len;

    auto data   = adp->data;
    auto d_data = adp->d_data;
    auto m      = adp->m;
    auto mxm    = adp->mxm;

    auto& wf       = ap->szwf;
    auto& subfiles = ap->subfiles;

    if (wf.lossy_dryrun) {
        LogAll(log_info, "invoke dry-run");

        if (ap->ndim == 1) {
            LorenzoNdConfig<1, Data, workflow::zip> lc(ap->dim4, ap->stride4, ap->nblk4, ap->radius, ap->eb);
            dr::lorenzo_1d1l<Data><<<lc.cfg.Dg, lc.cfg.Db, lc.cfg.Ns, lc.cfg.S>>>(lc.r_ctx, d_data);
        }
        else if (ap->ndim == 2) {
            LorenzoNdConfig<2, Data, workflow::zip> lc(ap->dim4, ap->stride4, ap->nblk4, ap->radius, ap->eb);
            dr::lorenzo_2d1l<Data><<<lc.cfg.Dg, lc.cfg.Db, lc.cfg.Ns, lc.cfg.S>>>(lc.r_ctx, d_data);
        }
        else if (ap->ndim == 3) {
            LorenzoNdConfig<3, Data, workflow::zip> lc(ap->dim4, ap->stride4, ap->nblk4, ap->radius, ap->eb);
            dr::lorenzo_3d1l<Data><<<lc.cfg.Dg, lc.cfg.Db, lc.cfg.Ns, lc.cfg.S>>>(lc.r_ctx, d_data);
        }
        HANDLE_ERROR(hipDeviceSynchronize());

        auto data_lossy = new Data[len]();
        hipMemcpy(data_lossy, d_data, len * sizeof(Data), hipMemcpyDeviceToHost);

        analysis::VerifyData<Data>(&ap->stat, data_lossy, data, len);
        analysis::PrintMetrics<Data>(&ap->stat, false, ap->eb, 0);

        hipHostFree(data);
        hipFree(d_data);
        exit(0);
    }
    LogAll(log_info, "invoke zipping");

    auto d_q = mem::CreateCUDASpace<Quant>(len);  // quant. code is not needed for dry-run

    // prediction-quantization
    {
        if (ap->ndim == 1) {
            LorenzoNdConfig<1, Data, workflow::zip> lc(ap->dim4, ap->stride4, ap->nblk4, ap->radius, ap->eb);
            fm::c_lorenzo_1d1l<Data, Quant><<<lc.cfg.Dg, lc.cfg.Db, lc.cfg.Ns, lc.cfg.S>>>(lc.z_ctx, d_data, d_q);
        }
        else if (ap->ndim == 2) {
            LorenzoNdConfig<2, Data, workflow::zip> lc(ap->dim4, ap->stride4, ap->nblk4, ap->radius, ap->eb);
            fm::c_lorenzo_2d1l<Data, Quant><<<lc.cfg.Dg, lc.cfg.Db, lc.cfg.Ns, lc.cfg.S>>>(lc.z_ctx, d_data, d_q);
        }
        else if (ap->ndim == 3) {
            LorenzoNdConfig<3, Data, workflow::zip> lc(ap->dim4, ap->stride4, ap->nblk4, ap->radius, ap->eb);
            fm::c_lorenzo_3d1l<Data, Quant><<<lc.cfg.Dg, lc.cfg.Db, lc.cfg.Ns, lc.cfg.S>>>(lc.z_ctx, d_data, d_q);
        }
        HANDLE_ERROR(hipDeviceSynchronize());
    }

    ::cusz::impl::PruneGatherAsCSR(d_data, mxm, m /*lda*/, m /*m*/, m /*n*/, nnz_outlier, &subfiles.c_fo_outlier);

    auto fmt_nnz = "(" + std::to_string(nnz_outlier / 1.0 / len * 100) + "%)";
    LogAll(log_info, "nnz/#outlier:", nnz_outlier, fmt_nnz, "saved");
    hipFree(d_data);  // ad-hoc, release memory for large dataset

    Quant* q;
    if (wf.skip_huffman_enc) {
        q = mem::CreateHostSpaceAndMemcpyFromDevice(d_q, len);
        io::WriteArrayToBinary(subfiles.c_fo_q, q, len);

        LogAll(log_info, "to store quant.code directly (Huffman enc skipped)");

        return;
    }

    // autotuning Huffman chunksize
    // subject to change, current `8*` is close to but may note deterministically optimal
    if (wf.autotune_huffman_chunk) {  //
        auto optimal_chunksize = 1;
        auto cuda_core_num     = cusz::tune::GetCUDACoreNum();
        auto cuda_thread_num   = 8 * cuda_core_num;  // empirical value

        while (optimal_chunksize * cuda_thread_num < len) optimal_chunksize *= 2;
        ap->huffman_chunk = optimal_chunksize;
    }

    if (wf.exp_partitioning_imbalance) {
        // 3D only
        auto part0     = ap->part4._0;
        auto part1     = ap->part4._1;
        auto part2     = ap->part4._2;
        auto num_part0 = (ap->dim4._0 - 1) / part0 + 1;
        auto num_part1 = (ap->dim4._1 - 1) / part1 + 1;
        auto num_part2 = (ap->dim4._2 - 1) / part2 + 1;

        LogAll(log_dbg, "p0:", ap->part4._0, " p1:", ap->part4._1, " p2:", ap->part4._2);
        LogAll(log_dbg, "num_part0:", num_part0, " num_part1:", num_part1, " num_part2:", num_part2);

        size_t block_stride1 = ap->part4._0, block_stride2 = block_stride1 * ap->part4._0;

        LogAll(log_dbg, "stride1:", ap->stride4._1, " stride2:", ap->stride4._2);
        LogAll(log_dbg, "blockstride1:", block_stride1, " blockstride2:", block_stride2);

        auto buffer_size = part0 * part1 * part2;
        LogAll(log_dbg, "buffer size:", buffer_size);
        auto quant_buffer = new Quant[buffer_size]();

        hipFree(d_data);
        hipHostFree(data);

        q = mem::CreateHostSpaceAndMemcpyFromDevice(d_q, len);
        hipFree(d_q);

        Index<3>::idx_t part_dims{part0, part1, part2};
        Index<3>::idx_t block_strides{1, (int)block_stride1, (int)block_stride2};
        Index<3>::idx_t global_strides{1, (int)ap->stride4._1, (int)ap->stride4._2};

        for (auto pk = 0; pk < num_part2; pk++) {
            for (auto pj = 0; pj < num_part1; pj++) {
                for (auto pi = 0; pi < num_part0; pi++) {
                    auto start = pk * part2 * ap->stride4._2 + pj * part1 * ap->stride4._1 + pi * part0;
                    CopyToBuffer_3D(quant_buffer, q, start, part_dims, block_strides, global_strides);
                    lossless::interface::HuffmanEncodeWithTree_3D<Quant, Huff>(
                        Index<3>::idx_t{pi, pj, pk}, subfiles.c_huff_base, quant_buffer, buffer_size, ap->dict_size);
                }
            }
        }

        delete[] quant_buffer;
        delete[] q;

        exit(0);
    }

    std::tie(num_bits, num_uints, huff_meta_size, nvcomp_in_use) = lossless::interface::HuffmanEncode<Quant, Huff>(
        subfiles.c_huff_base, d_q, len, ap->huffman_chunk, wf.lossless_nvcomp_cascade, ap->dict_size,
        wf.exp_export_codebook);

    LogAll(log_dbg, "to store Huffman encoded quant.code (default)");

    hipFree(d_q);
}

template <bool If_FP, int DataByte, int QuantByte, int HuffByte>
void cusz::interface::Decompress(
    argpack* ap,
    int&     nnz_outlier,
    size_t&  total_bits,
    size_t&  total_uint,
    size_t&  huffman_metadata_size,
    bool     nvcomp_in_use)
{
    using Data  = typename DataTrait<If_FP, DataByte>::Data;
    using Quant = typename QuantTrait<QuantByte>::Quant;
    using Huff  = typename HuffTrait<HuffByte>::Huff;

    auto& wf       = ap->szwf;
    auto& subfiles = ap->subfiles;

    auto m   = ::cusz::impl::GetEdgeOfReinterpretedSquare(ap->len);
    auto mxm = m * m;

    LogAll(log_info, "invoke unzip");

    Quant* xq;
    // step 1: read from filesystem or do Huffman decoding to get quant code
    if (wf.skip_huffman_enc) {
        LogAll(log_info, "load quant.code from filesystem");
        xq = io::ReadBinaryToNewArray<Quant>(subfiles.x_fi_q, ap->len);
    }
    else {
        LogAll(log_info, "Huffman decode -> quant.code");
        xq = lossless::interface::HuffmanDecode<Quant, Huff>(
            subfiles.cx_path2file, ap->len, ap->huffman_chunk, total_uint, nvcomp_in_use, ap->dict_size);
        if (wf.verify_huffman) {
            LogAll(log_warn, "Verifying Huffman is temporarily disabled in this version (2021 Week 3");
            /*
            // TODO check in argpack
            if (subfiles.x_fi_origin == "") {
                cerr << log_err << "use \"--origin /path/to/origin_data\" to specify the original datum." << endl;
                exit(-1);
            }
            cout << log_info << "Verifying Huffman codec..." << endl;
            ::cusz::impl::VerifyHuffman<Data, Quant>(subfiles.x_fi_origin, len, xq, ap->huffman_chunk, dims,
            eb_variants);
             */
        }
    }
    auto d_xq = mem::CreateDeviceSpaceAndMemcpyFromHost(xq, ap->len);

    auto d_outlier = mem::CreateCUDASpace<Data>(mxm);
    ::cusz::impl::ScatterFromCSR<Data>(
        d_outlier, mxm, m /*lda*/, m /*m*/, m /*n*/, &nnz_outlier, &subfiles.x_fi_outlier);

    // TODO merge d_outlier and d_data
    auto d_xdata = mem::CreateCUDASpace<Data>(ap->len);

    {
        // temporary
        if (ap->ndim == 1) {
            LorenzoNdConfig<1, Data, workflow::unzip> lc(ap->dim4, ap->stride4, ap->nblk4, ap->radius, ap->eb);
            fm::x_lorenzo_1d1l<Data, Quant>
                <<<lc.cfg.Dg, lc.cfg.Db, lc.cfg.Ns, lc.cfg.S>>>(lc.x_ctx, d_xdata, d_outlier, d_xq);
        }
        else if (ap->ndim == 2) {
            LorenzoNdConfig<2, Data, workflow::unzip> lc(ap->dim4, ap->stride4, ap->nblk4, ap->radius, ap->eb);
            fm::x_lorenzo_2d1l<Data, Quant>
                <<<lc.cfg.Dg, lc.cfg.Db, lc.cfg.Ns, lc.cfg.S>>>(lc.x_ctx, d_xdata, d_outlier, d_xq);
        }
        else if (ap->ndim == 3) {
            LorenzoNdConfig<3, Data, workflow::unzip> lc(ap->dim4, ap->stride4, ap->nblk4, ap->radius, ap->eb);
            fm::x_lorenzo_3d1l<Data, Quant>
                <<<lc.cfg.Dg, lc.cfg.Db, lc.cfg.Ns, lc.cfg.S>>>(lc.x_ctx, d_xdata, d_outlier, d_xq);
        }
        HANDLE_ERROR(hipDeviceSynchronize());
    }

    auto xdata = mem::CreateHostSpaceAndMemcpyFromDevice(d_xdata, ap->len);

    LogAll(log_info, "reconstruct error-bounded datum");

    size_t archive_bytes = 0;
    // TODO huffman chunking metadata
    if (not wf.skip_huffman_enc)
        archive_bytes += total_uint * sizeof(Huff)  // Huffman coded
                         + huffman_metadata_size;   // chunking metadata and reverse codebook
    else
        archive_bytes += ap->len * sizeof(Quant);
    archive_bytes += nnz_outlier * (sizeof(Data) + sizeof(int)) + (m + 1) * sizeof(int);

    // TODO g++ and clang++ use mangled type_id name, add macro
    // https://stackoverflow.com/a/4541470/8740097
    auto demangle = [](const char* name) {
        int   status = -4;
        char* res    = abi::__cxa_demangle(name, nullptr, nullptr, &status);

        const char* const demangled_name = (status == 0) ? res : name;
        string            ret_val(demangled_name);
        free(res);
        return ret_val;
    };

    if (wf.skip_huffman_enc) {
        cout << log_info << "dtype is \""         //
             << demangle(typeid(Data).name())     // demangle
             << "\", and quant. code type is \""  //
             << demangle(typeid(Quant).name())    // demangle
             << "\"; a CR of no greater than "    //
             << (sizeof(Data) / sizeof(Quant)) << " is expected when Huffman codec is skipped." << endl;
    }

    if (wf.pre_binning) cout << log_info << "Because of 2x2->1 binning, extra 4x CR is added." << endl;

    // TODO move CR out of VerifyData
    if (subfiles.x_fi_origin != "") {
        LogAll(log_info, "load the original datum for comparison");

        auto odata = io::ReadBinaryToNewArray<Data>(subfiles.x_fi_origin, ap->len);
        analysis::VerifyData(&ap->stat, xdata, odata, ap->len);
        analysis::PrintMetrics<Data>(&ap->stat, false, ap->eb, archive_bytes, wf.pre_binning ? 4 : 1);

        delete[] odata;
    }
    LogAll(log_info, "output:", subfiles.cx_path2file + ".szx");

    if (wf.skip_write_output)
        io::WriteArrayToBinary(subfiles.x_fo_xd, xdata, ap->len);
    else {
        LogAll(log_dbg, "skipped writing unzipped to filesystem");
    }

    // clean up
    delete[] xdata;
    delete[] xq;
    hipFree(d_xdata);
    hipFree(d_outlier);
    hipFree(d_xq);
}

typedef struct DataPack<float> adp_f32_t;
namespace szin = cusz::interface;

// TODO top-level instantiation really reduce compilation time?
// clang-format off
template void szin::Compress<true, 4, 1, 4>(argpack*, adp_f32_t*, int&, size_t&, size_t&, size_t&, bool&);
template void szin::Compress<true, 4, 1, 8>(argpack*, adp_f32_t*, int&, size_t&, size_t&, size_t&, bool&);
template void szin::Compress<true, 4, 2, 4>(argpack*, adp_f32_t*, int&, size_t&, size_t&, size_t&, bool&);
template void szin::Compress<true, 4, 2, 8>(argpack*, adp_f32_t*, int&, size_t&, size_t&, size_t&, bool&);

template void szin::Decompress<true, 4, 1, 4>(argpack*, int&, size_t&, size_t&, size_t&, bool);
template void szin::Decompress<true, 4, 1, 8>(argpack*, int&, size_t&, size_t&, size_t&, bool);
template void szin::Decompress<true, 4, 2, 4>(argpack*, int&, size_t&, size_t&, size_t&, bool);
template void szin::Decompress<true, 4, 2, 8>(argpack*, int&, size_t&, size_t&, size_t&, bool);
